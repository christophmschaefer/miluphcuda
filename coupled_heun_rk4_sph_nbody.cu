#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2020 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */



/* coupled heun and rk4 integrator */
// Heun is used for the SPH particles
// RK4 for the NBODYs
// following the idea from Daniel Thun in Append A of https://www.aanda.org/articles/aa/pdf/2018/08/aa32804-18.pdf
// note: this integrator is designed to calculate the nbody orbit with high precision while keeping
//       the hydro computational effort low (meaning with a higher timestep)
// designed for circumbinary disks


// authors: Evita Vavilina and cms



#include "coupled_heun_rk4_sph_nbody.h"
#include "config_parameter.h"
#include "timeintegration.h"
#include "parameter.h"
#include "memory_handling.h"
#include "miluph.h"
#include "pressure.h"
#include "rhs.h"
#include "gravity.h"
#include "damage.h"
#include <float.h>


extern __device__ double endTimeD, currentTimeD;
extern __device__ double substep_currentTimeD;
extern __device__ double dt;
extern __device__ double dtmax;
extern __device__ int blockCount;
extern __device__ double emin_d;
extern __device__ double Smin_d;
extern __device__ double rhomin_d;
extern __device__ double damagemin_d;
extern __device__ double alphamin_d;
extern __device__ double betamin_d;
extern __device__ double alpha_epspormin_d;
extern __device__ double epsilon_vmin_d;
extern __device__ int pressureChangeSmallEnough;
extern __device__ double maxpressureDiff;


extern double L_ini;


__global__ void CorrectorStep_heun()
{
    register int i;
#if SOLID
    register int j;
    register int k;
#endif

    // particle loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        p.x[i] = p.x[i] + dt/2 * (predictor.dxdt[i] + p.dxdt[i]);
#if DIM > 1
        p.y[i] = p.y[i] + dt/2 * (predictor.dydt[i] + p.dydt[i]);
        p.vy[i] = p.vy[i] + dt/2 * (predictor.ay[i] + p.ay[i]);
        p.ay[i] = 0.5*(predictor.ay[i] + p.ay[i]);
#endif
        p.vx[i] = p.vx[i] + dt/2 * (predictor.ax[i] + p.ax[i]);
        p.ax[i] = 0.5*(predictor.ax[i] + p.ax[i]);
#if DIM == 3
        p.z[i] = p.z[i] + dt/2 * (predictor.dzdt[i] + p.dzdt[i]);
        p.vz[i] = p.vz[i] + dt/2 * (predictor.az[i] + p.az[i]);
        p.az[i] = 0.5*(predictor.az[i] + p.az[i]);
#endif
#if INTEGRATE_ENERGY
        p.e[i] = p.e[i] + dt/2 * (predictor.dedt[i] + p.dedt[i]);
        p.dedt[i] = 0.5*(predictor.dedt[i] + p.dedt[i]);
#endif
#if FRAGMENTATION
        p.d[i] = p.d[i] + dt/2 * (predictor.dddt[i] + p.dddt[i]);
        p.dddt[i] = 0.5*(predictor.dddt[i] + p.dddt[i]);
#endif
#if INTEGRATE_SML
        p.h[i] = p.h[i] + dt/2 * (predictor.dhdt[i] + p.dhdt[i]);
        p.dhdt[i] = 0.5 * (predictor.dhdt[i] + p.dhdt[i]);
#else
        p.h[i] = predictor.h[i];
#endif
#if JC_PLASTICITY
        p.ep[i] = p.ep[i] + dt/2 * (predictor.edotp[i] + p.edotp[i]);
        p.edotp[i] = 0.5*(predictor.edotp[i] + p.edotp[i]);
        p.T[i] = p.T[i] + dt/2 * (predictor.dTdt[i] + p.dTdt[i]);
        p.dTdt[i] = 0.5*(predictor.dTdt[i] + p.dTdt[i]);
#endif
#if FRAGMENTATION
# if PALPHA_POROSITY
//        if (p.drhodt[i] > 0 && predictor.p[i] > predictor.pold[i]) {
		if (predictor.p[i] > predictor.pold[i]) {
            p.damage_porjutzi[i] = p.damage_porjutzi[i] + dt/2 *(predictor.ddamage_porjutzidt[i] + p.ddamage_porjutzidt[i]);
        } else {
            p.d[i] = p.d[i];
            p.damage_porjutzi[i] = p.damage_porjutzi[i];
        }
# endif
        p.numActiveFlaws[i] = predictor.numActiveFlaws[i];
#endif
        // change drhodt after PALPHA_POROSITY
#if INTEGRATE_DENSITY
        p.rho[i] = p.rho[i] + dt/2 * (predictor.drhodt[i] + p.drhodt[i]);
        p.drhodt[i] = 0.5*(predictor.drhodt[i]+ p.drhodt[i]);
#else
        p.rho[i] = p.rho[i];
#endif
#if SIRONO_POROSITY
        p.rho_0prime[i] = p.rho_0prime[i];
        p.rho_c_plus[i] = p.rho_c_plus[i];
        p.rho_c_minus[i] = p.rho_c_minus[i];
        p.compressive_strength[i] = p.compressive_strength[i];
        p.tensile_strength[i] = p.tensile_strength[i];
        p.shear_strength[i] = p.shear_strength[i];
        p.K[i] = p.K[i];
        p.flag_rho_0prime[i] = p.flag_rho_0prime[i];
        p.flag_plastic[i] = p.flag_plastic[i];
#endif
#if EPSALPHA_POROSITY
        p.alpha_epspor[i] = p.alpha_epspor[i] + dt/2 * (predictor.dalpha_epspordt[i] + p.dalpha_epspordt[i]);
        p.dalpha_epspordt[i] = 0.5*(predictor.dalpha_epspordt[i] + p.dalpha_epspordt[i]);
        p.epsilon_v[i] = p.epsilon_v[i] + dt/2 * (predictor.depsilon_vdt[i] + p.depsilon_vdt[i]);
        p.depsilon_vdt[i] = 0.5*(predictor.depsilon_vdt[i] + p.depsilon_vdt[i]);
#endif
#if INVISCID_SPH
        p.beta[i] = p.beta[i] + dt/2 * (predictor.dbetadt[i] + p.dbetadt[i]);
        p.dbetadt[i] = 0.5 * (predictor.dbetadt[i] + p.dbetadt[i]);
#endif
#if SOLID
        for (j = 0; j < DIM; j++) {
            for (k = 0; k < DIM; k++) {
                p.S[stressIndex(i,j,k)] = p.S[stressIndex(i,j,k)] + dt/2  *
                    (predictor.dSdt[stressIndex(i,j,k)] + p.dSdt[stressIndex(i,j,k)]);
                p.dSdt[stressIndex(i,j,k)] = 0.5*(predictor.dSdt[stressIndex(i,j,k)] +
                        p.dSdt[stressIndex(i,j,k)]);
            }
        }
#if PALPHA_POROSITY
        /* check if we have compaction and change alpha accordingly */
//        if (p.drhodt[i] > 0 && predictor.p[i] > predictor.pold[i]) {
		if (predictor.p[i] > predictor.pold[i]) {
            p.alpha_jutzi[i] = p.alpha_jutzi[i] + dt/2 * ( p.dalphadt[i] + predictor.dalphadt[i]);
        } else {
            p.alpha_jutzi[i] = p.alpha_jutzi[i];
        }
        // next line, Oli & Christoph personal communication 2019-06-24
        p.p[i] = predictor.p[i];
        p.pold[i] = predictor.pold[i];
        p.alpha_jutzi_old[i] = p.alpha_jutzi_old[i];
#endif
#endif
    }
}

__global__ void PredictorStep_heun()
{
    register int i;
#if SOLID
    register int j;
    register int k;
#endif

    // particle loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        predictor.x[i] = p.x[i] + dt * p.dxdt[i];
        predictor.vx[i] = p.vx[i] + dt * p.ax[i];

#if DIM > 1
        predictor.y[i] = p.y[i] + dt * p.dydt[i];
        predictor.vy[i] = p.vy[i] + dt * p.ay[i];
#endif
#if DIM > 2
        predictor.z[i] = p.z[i] + dt * p.dzdt[i];
        predictor.vz[i] = p.vz[i] + dt * p.az[i];
#endif
#if INTEGRATE_DENSITY
        predictor.rho[i] = p.rho[i] + dt * p.drhodt[i];
#else
        predictor.rho[i] = p.rho[i];
#endif
        predictor.drhodt[i] = p.drhodt[i];
#if INTEGRATE_ENERGY
        predictor.e[i] = p.e[i] + dt * p.dedt[i];
#endif

#if INTEGRATE_SML
        predictor.h[i] = p.h[i] + dt * p.dhdt[i];
#else
        predictor.h[i] = p.h[i];
#endif

#if PALPHA_POROSITY
        predictor.alpha_jutzi[i] = p.alpha_jutzi[i] + dt * p.dalphadt[i];
        predictor.alpha_jutzi_old[i] = p.alpha_jutzi_old[i];
#endif

#if EPSALPHA_POROSITY
        predictor.alpha_epspor[i] = p.alpha_epspor[i] + dt * p.dalpha_epspordt[i];
        predictor.epsilon_v[i] = p.epsilon_v[i] + dt * p.depsilon_vdt[i];
#endif

#if FRAGMENTATION
        predictor.d[i] = p.d[i] + dt * p.dddt[i];
        predictor.numActiveFlaws[i] = p.numActiveFlaws[i];
# if PALPHA_POROSITY
        predictor.damage_porjutzi[i] = p.damage_porjutzi[i] + dt * p.ddamage_porjutzidt[i];
        // cms 2019-06-25
        // pressuremaxchange function will compare predictor.p - predictor.pold and
        // calc_pressure in integrator will overwrite predictor.p to the value *after* the test time
        // step
        //predictor.p[i] = p.p[i];
        predictor.pold[i] = p.p[i];
# endif
#endif
#if SIRONO_POROSITY
        predictor.rho_0prime[i] = p.rho_0prime[i];
        predictor.rho_c_plus[i] = p.rho_c_plus[i];
        predictor.rho_c_minus[i] = p.rho_c_minus[i];
        predictor.compressive_strength[i] = p.compressive_strength[i];
        predictor.tensile_strength[i] = p.tensile_strength[i];
        predictor.shear_strength[i] = p.shear_strength[i];
        predictor.K[i] = p.K[i];
        predictor.flag_rho_0prime[i] = p.flag_rho_0prime[i];
        predictor.flag_plastic[i] = p.flag_plastic[i];
#endif
#if JC_PLASTICITY
        predictor.ep[i] = p.ep[i] + dt * p.edotp[i];
        predictor.T[i] = p.T[i] + dt * p.dTdt[i];
#endif
#if INVISCID_SPH
        predictor.beta[i] = p.beta[i] + dt * p.dbetadt[i];
#endif
#if SOLID
        for (j = 0; j < DIM; j++) {
            for (k = 0; k < DIM; k++) {
                predictor.S[stressIndex(i,j,k)] = p.S[stressIndex(i,j,k)] + dt * p.dSdt[stressIndex(i,j,k)];
            }
        }
#endif
    }

}

#if PALPHA_POROSITY
/* check the pressure change to avoid large deviation from the crush-curve */
__global__ void pressureChangeCheck_heun(double *maxpressureDiffPerBlock)
{
    __shared__ double sharedMaxpressureDiff[NUM_THREADS_PC_INTEGRATOR];
    double localMaxpressureDiff = 0.0;
    double tmp = 0;
    int i, j, k, m;
    maxpressureDiff = 0.0;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        if (matEOS[p_rhs.materialId[i]] == EOS_TYPE_JUTZI || matEOS[p_rhs.materialId[i]] == EOS_TYPE_JUTZI_MURNAGHAN || matEOS[p_rhs.materialId[i]] == EOS_TYPE_JUTZI_ANEOS) {
        /* when this function is called, p points to predictor_device */
            // only consider pressure if the particle is compressed
            if (p.drhodt[i] > 0) {
                tmp = (p.p[i] - p.pold[i]);
            } else {
                tmp = 0;
            }
            localMaxpressureDiff = max(tmp, localMaxpressureDiff);
        }
    }
    i = threadIdx.x;
    sharedMaxpressureDiff[i] = localMaxpressureDiff;
    for (j = NUM_THREADS_PC_INTEGRATOR / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sharedMaxpressureDiff[i] = localMaxpressureDiff = max(localMaxpressureDiff, sharedMaxpressureDiff[k]);
        }
    }
    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        maxpressureDiffPerBlock[k] = localMaxpressureDiff;
        m = gridDim.x - 1;
        if (m == atomicInc((unsigned int *)&blockCount, m)) {
            // last block, so combine all block results
            for (j = 0; j <= m; j++) {
                localMaxpressureDiff = max(localMaxpressureDiff, maxpressureDiffPerBlock[j]);
            }
            maxpressureDiff = localMaxpressureDiff;
            // reset block count
            blockCount = 0;
        }
        if (maxpressureDiff > max_abs_pressure_change) {
            printf("%e\n", maxpressureDiff);
            pressureChangeSmallEnough = FALSE;
            dt = 0.25 * dt;
            dt = min(dt, endTimeD - currentTimeD);
        } else {
            pressureChangeSmallEnough = TRUE;
        }
    }

}
#endif

__global__ void setTimestep_heun(double *forcesPerBlock, double *courantPerBlock, double *dtSPerBlock, double *dtePerBlock, double *dtrhoPerBlock, double *dtdamagePerBlock, double *dtalphaPerBlock, double *dtartviscPerBlock, double *dtbetaPerBlock, double *dtalpha_epsporPerBlock, double *dtepsilon_vPerBlock)
{

#define SAFETY_FIRST 0.1

    __shared__ double sharedForces[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedCourant[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedArtVisc[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedS[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharede[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedrho[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double shareddamage[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedalpha[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedbeta[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedalpha_epspor[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedepsilon_v[NUM_THREADS_LIMITTIMESTEP];

    int i, j, k, m;
    int d, dd;
    int index;
    int hasEnergy;
    double forces = DBL_MAX, courant = DBL_MAX;
    double dtx = DBL_MAX;
    double dtS = DBL_MAX;
    double dtrho = DBL_MAX;
    double dte = DBL_MAX;
    double dtdamage = DBL_MAX;
    double dtalpha = DBL_MAX;
    double dtbeta = DBL_MAX;
    double dtalpha_epspor = DBL_MAX;
    double dtepsilon_v = DBL_MAX;
    double temp;
    double sml;
    int matId;
#if SOLID
    double myS, dS;
#endif
    double ax, ay;
#if DIM == 3
    double az;
#endif
    double dtartvisc = DBL_MAX;

    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        matId = p_rhs.materialId[i];
#if INTEGRATE_ENERGY
        hasEnergy = 0;

        switch  (matEOS[matId]) {
            case (EOS_TYPE_TILLOTSON):
                hasEnergy = 1;
                break;
            case (EOS_TYPE_JUTZI):
                hasEnergy = 1;
                break;
			case (EOS_TYPE_JUTZI_ANEOS):
				hasEnergy = 1;
				break;
            case (EOS_TYPE_SIRONO):
                hasEnergy = 1;
                break;
            case (EOS_TYPE_EPSILON):
                hasEnergy = 1;
                break;
            case (EOS_TYPE_ANEOS):
                hasEnergy = 1;
                break;
            default:
                hasEnergy = 0;
                break;
        }
#endif
        ax = p.ax[i];
#if DIM > 1
        ay = p.ay[i];
#endif
#if DIM == 3
        az = p.az[i];
#endif
        temp = ax*ax;
#if DIM > 1
        temp += + ay*ay;
#endif
#if DIM == 3
        temp += az*az;
#endif

        sml = p.h[i];
        temp = sqrt(sml / sqrt(temp));
        forces = min(forces, temp);
        temp = sml / p.cs[i];
        courant = min(courant, temp);

#if ARTIFICIAL_VISCOSITY
        temp = COURANT_FACT * sml / (p.cs[i] + 1.2 * (matAlpha[matId]) * p.cs[i] + matBeta[matId] * p.muijmax[i]);
        dtartvisc = min(dtartvisc, temp);
#endif
#if INVISCID_SPH
        if (p.dbetadt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.beta[i])+betamin_d)/fabs(p.dbetadt[i]);
            dtbeta = min(temp, dtbeta);
        }

#endif
#if SOLID
        myS = 0;
        dS = 0;

        for (d = 0; d < DIM; d++) {
            for (dd = 0; dd < DIM; dd++) {
                index = i*DIM*DIM+d*DIM+dd;
                myS = fabs(p.S[index]);
                dS = fabs(p.dSdt[index]);
                if (dS > 0) {
                    temp = SAFETY_FIRST * (myS+Smin_d)/dS;
                    dtS = min(temp, dtS);
                }
            }
        }
#endif
#if INTEGRATE_DENSITY
        if (p.drhodt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.rho[i])+rhomin_d)/fabs(p.drhodt[i]);
            dtrho = min(temp, dtrho);
        }
#endif
#if INTEGRATE_ENERGY
        if (p.dedt[i] != 0 && hasEnergy) {
            temp = SAFETY_FIRST * (fabs(p.e[i])+emin_d)/fabs(p.dedt[i]);
            dte = min(temp, dte);
        }
#endif

#if PALPHA_POROSITY
        if (p.dalphadt[i] != 0) {
            temp = 1.0e-2 / fabs(p.dalphadt[i]);
            dtalpha = min(temp, dtalpha);
        }
#endif

#if EPSALPHA_POROSITY
        if (p.dalpha_epspordt[i] != 0) {
            temp = 1.0e-1 / fabs(p.dalpha_epspordt[i]);
            dtalpha_epspor = min(temp, dtalpha_epspor);
        }

        if (p.depsilon_vdt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.epsilon_v[i])+epsilon_vmin_d)/fabs(p.depsilon_vdt[i]);
            dtepsilon_v = min(temp, dtepsilon_v);
        }
#endif

#if FRAGMENTATION
        if (p.dddt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.d[i])+damagemin_d)/fabs(p.dddt[i]);
            dtdamage = min(temp, dtdamage);
        }
#endif
    }

    i = threadIdx.x;
    sharedForces[i] = forces;
    sharedCourant[i] = courant;
    sharedS[i] = dtS;
    sharede[i] = dte;
    sharedrho[i] = dtrho;
    shareddamage[i] = dtdamage;
    sharedalpha[i] = dtalpha;
    sharedalpha_epspor[i] = dtalpha_epspor;
    sharedepsilon_v[i] = dtepsilon_v;
#if ARTIFICIAL_VISCOSITY
    sharedArtVisc[i] = dtartvisc;
#endif
#if INVISCID_SPH
    sharedbeta[i] = dtbeta;
#endif
    for (j = NUM_THREADS_LIMITTIMESTEP / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sharedForces[i] = forces = min(forces, sharedForces[k]);
            sharedCourant[i] = courant = min(courant, sharedCourant[k]);
            sharedS[i] = dtS = min(dtS, sharedS[k]);
            sharede[i] = dte = min(dte, sharede[k]);
            sharedrho[i] = dtrho = min(dtrho, sharedrho[k]);
            shareddamage[i] = dtdamage = min(dtdamage, shareddamage[k]);
            sharedalpha[i] = dtalpha = min(dtalpha, sharedalpha[k]);
            sharedalpha_epspor[i] = dtalpha_epspor = min(dtalpha_epspor, sharedalpha_epspor[k]);
            sharedepsilon_v[i] = dtepsilon_v = min(dtepsilon_v, sharedepsilon_v[k]);
#if ARTIFICIAL_VISCOSITY
            sharedArtVisc[i] = dtartvisc = min(dtartvisc, sharedArtVisc[k]);
#endif
#if INVISCID_SPH
            sharedbeta[i] = dtbeta = min(dtbeta, sharedbeta[k]);
#endif
        }
    }
    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        forcesPerBlock[k] = forces;
        courantPerBlock[k] = courant;
        dtSPerBlock[k] = dtS;
        dtePerBlock[k] = dte;
        dtrhoPerBlock[k] = dtrho;
        dtdamagePerBlock[k] = dtdamage;
        dtalphaPerBlock[k] = dtalpha;
        dtalpha_epsporPerBlock[k] = dtalpha_epspor;
        dtepsilon_vPerBlock[k] = dtepsilon_v;
#if ARTIFICIAL_VISCOSITY
        dtartviscPerBlock[k] = dtartvisc;
#endif
#if INVISCID_SPH
        dtbetaPerBlock[k] = dtbeta;
#endif
        m = gridDim.x - 1;
        if (m == atomicInc((unsigned int *)&blockCount, m)) {
            // last block, so combine all block results
            for (j = 0; j <= m; j++) {
                forces = min(forces, forcesPerBlock[j]);
                courant = min(courant, courantPerBlock[j]);
                dtS = min(dtS, dtSPerBlock[j]);
                dte = min(dte, dtePerBlock[j]);
                dtrho = min(dtrho, dtrhoPerBlock[j]);
                dtdamage = min(dtdamage, dtdamagePerBlock[j]);
                dtalpha = min(dtalpha, dtalphaPerBlock[j]);
                dtalpha_epspor = min(dtalpha_epspor, dtalpha_epsporPerBlock[j]);
                dtepsilon_v = min(dtepsilon_v, dtepsilon_vPerBlock[j]);
#if ARTIFICIAL_VISCOSITY
                dtartvisc = min(dtartvisc, dtartviscPerBlock[j]);
#endif
#if INVISCID_SPH
                dtbeta = min(dtbeta, dtbetaPerBlock[j]);
#endif
            }
            // set new timestep
            dt = dtx = min(COURANT_FACT*courant, FORCES_FACT*forces);
#if SOLID
            dt = min(dt, dtS);
#endif
#if INTEGRATE_ENERGY
            dt = min(dt, dte);
#endif
#if INTEGRATE_DENSITY
            dt = min(dt, dtrho);
#endif
#if FRAGMENTATION
            dt = min(dt, dtdamage);
#endif
#if PALPHA_POROSITY
         //   dt = min(dt, dtalpha);
#endif
#if EPSALPHA_POROSITY
            dt = min(dt, dtalpha_epspor);
            dt = min(dt, dtepsilon_v);
#endif
#if ARTIFICIAL_VISCOSITY
            dt = min(dt, dtartvisc);
#endif
#if INVISCID_SPH
            dt = min(dt, dtbeta);
#endif
            dt = min(dt, endTimeD - currentTimeD);
            if (dt > dtmax) dt = dtmax;
            printf("Time Step Information: dt(v and x): %.17e ", dtx);
#if INTEGRATE_DENSITY
            printf("rho: %.17e ", dtrho);
#endif
#if INTEGRATE_ENERGY
            printf("e: %.17e ", dte);
#endif
#if SOLID
            printf("S: %.17e ", dtS);
#endif
#if FRAGMENTATION
            printf("damage: %.17e ", dtdamage);
#endif
#if PALPHA_POROSITY
            printf("distention: %.17e ", dtalpha);
#endif
#if EPSALPHA_POROSITY
            printf("distention: %.17e ", dtalpha_epspor);
            printf("epsilon: %.17e ", dtepsilon_v);
#endif
            printf("\n");
            printf("time: %.17e timestep set to %.17e, integrating until %.17e \n", currentTimeD, dt, endTimeD);
#if !PALPHA_POROSITY
            currentTimeD += dt;
#endif
			// reset block count
			blockCount = 0;
		}
	}
}




void heun_rk4()
{

    double *courantPerBlock, *forcesPerBlock;
    double *dtSPerBlock, *dtePerBlock, *dtrhoPerBlock;
    double *dtdamagePerBlock;
    double *dtalphaPerBlock;
    double *dtbetaPerBlock;
    double *dtartviscPerBlock;
    double *dtalpha_epsporPerBlock;
    double *dtepsilon_vPerBlock;
    double *maxpressureDiffPerBlock;
    int pressureChangeSmallEnough_host;


    cudaVerify(hipMalloc((void**)&courantPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&forcesPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtSPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtePerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtrhoPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtdamagePerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtalphaPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtbetaPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&maxpressureDiffPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtartviscPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtalpha_epsporPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtepsilon_vPerBlock, sizeof(double)*numberOfMultiprocessors));

    int lastTimestep = startTimestep + numberOfTimesteps;
    int timestep;
    double maxpressureDiff_host;
    double maxpressureDiff_previous;
    int maxpressureDiff_cnt;
    double substep_currentTime;
    currentTime = startTime;
    double endTime = startTime;

    int allocate_immutables = 1;
    // alloc mem for one rhs
    allocate_particles_memory(&predictor_device, allocate_immutables);
    copy_particles_immutables_device_to_device(&predictor_device, &p_device);
    /* tell the gpu the current time */
    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(currentTimeD), &currentTime, sizeof(double)));
    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(predictor), &predictor_device, sizeof(struct Particle)));



#if GRAVITATING_POINT_MASSES
    int rkstep;
    int with_feedback;

    // alloc mem for multiple rhs and copy immutables
    for (rkstep = 0; rkstep < 4; rkstep++) {
        allocate_pointmass_memory(&rk4_pointmass_device[rkstep], allocate_immutables);
    }
    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(rk4_pointmass), &rk4_pointmass_device, sizeof(struct Pointmass) * 4));
#endif


    for (timestep = startTimestep; timestep < lastTimestep; timestep++) {
        fprintf(stdout, "calculating step %d\n", timestep);
        printf("\nstep %d / %d\n", timestep, lastTimestep);
        endTime += timePerStep;
        fprintf(stdout, " currenttime: %e \t endtime: %e\n", currentTime, endTime);
        /* tell the gpu the time step */
        if (timePerStep > param.maxtimestep) {
            fprintf(stdout, "timestep was larger than maxtimestep given by -M, reducing to %e\n", param.maxtimestep);
            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(dt), &param.maxtimestep, sizeof(double)));
        } else {
            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(dt), &timePerStep, sizeof(double)));
        }


        // checking for changes in angular momentum
        if (param.angular_momentum_check > 0) {
            double L_current = calculate_angular_momentum();
            double L_change_relative;
            if (L_ini > 0) {
                L_change_relative = fabs((L_ini - L_current)/L_ini);
            }
            if (param.verbose) {
                fprintf(stdout, "Checking angular momentum conservation.\n");
                fprintf(stdout, "Initial angular momentum of the particles: %.17e\n", L_ini);
                fprintf(stdout, "Current angular momentum of the particles: %.17e\n", L_current);
                fprintf(stdout, "Relative change: %.17e\n", L_change_relative);
            }
            if (L_change_relative > param.angular_momentum_check) {
                fprintf(stderr, "Conservation of angular momentum violated. Exiting.\n");
                exit(111);
            }
        }


        /* tell the gpu the end time */
        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(endTimeD), &endTime, sizeof(double)));

		while (currentTime < endTime) {
			cudaVerify(hipDeviceSynchronize());
			// calculate first right hand side with p_device
	        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &p_device, sizeof(struct Particle)));
            cudaVerify(hipDeviceSynchronize());
            cudaVerify(hipMemcpyFromSymbol(&currentTime, HIP_SYMBOL(currentTimeD), sizeof(double)));
            substep_currentTime = currentTime;
            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(substep_currentTimeD), &substep_currentTime, sizeof(double)));
#if GRAVITATING_POINT_MASSES
	        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
            rightHandSide();
            cudaVerify(hipDeviceSynchronize());
            cudaVerifyKernel((setTimestep_heun<<<numberOfMultiprocessors, NUM_THREADS_LIMITTIMESTEP>>>(
                              forcesPerBlock, courantPerBlock,
                              dtSPerBlock, dtePerBlock, dtrhoPerBlock, dtdamagePerBlock,
                              dtalphaPerBlock, dtartviscPerBlock, dtbetaPerBlock, dtalpha_epsporPerBlock, dtepsilon_vPerBlock)));
            cudaVerify(hipDeviceSynchronize());
            /* get the time and the time step from the gpu */
            cudaVerify(hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double)));
#if GRAVITATING_POINT_MASSES
            // calculate disk feedback if wanted
            backreaction_from_disk_to_point_masses(TRUE);
#endif
			cudaVerify(hipDeviceSynchronize());
            pressureChangeSmallEnough_host = FALSE;
            maxpressureDiff_cnt = 0;
            maxpressureDiff_host = 0;
            maxpressureDiff_previous = 0;
            while (pressureChangeSmallEnough_host == FALSE) {
	            // do the predictor step (writes to predictor)
                printf("First step with time step: %e at time: %e.\n", dt_host, currentTime);
	            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &p_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
                // fix this for rk4
	            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
    	        cudaVerifyKernel((PredictorStep_heun<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>()));
			    cudaVerify(hipDeviceSynchronize());


#define DISABLE_PRESSURE_CHECK_POROSITY_FOR_THE_TIME_BEING 0


#if !DISABLE_PRESSURE_CHECK_POROSITY_FOR_THE_TIME_BEING
#if PALPHA_POROSITY
                /* check if the step was too large */
                /* check the pressure at predictor step */
		        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &predictor_device, sizeof(struct Particle)));
				cudaVerifyKernel((calculatePressure<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    			cudaVerify(hipDeviceSynchronize());
			    cudaVerify(hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double)));
				printf("before pressure change check: dt_host: %e\n", dt_host);
				cudaVerifyKernel((pressureChangeCheck_heun<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>(maxpressureDiffPerBlock)));
    			cudaVerify(hipDeviceSynchronize());
                cudaVerify(hipMemcpyFromSymbol(&pressureChangeSmallEnough_host, HIP_SYMBOL(pressureChangeSmallEnough), sizeof(int)));
                cudaVerify(hipMemcpyFromSymbol(&maxpressureDiff_host, HIP_SYMBOL(maxpressureDiff), sizeof(double)));
#else
                pressureChangeSmallEnough_host = TRUE;
#endif // PALPHA_POROSITY
#else
                pressureChangeSmallEnough_host = TRUE;
#endif // !DISABLE_PRESSURE_CHECK_POROSITY_FOR_THE_TIME_BEING


			    cudaVerify(hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double)));
				printf("after pressure change check: dt_host: %e\n", dt_host);
				printf("pressureChangeSmallEnough_host: %d\n", pressureChangeSmallEnough_host);
				if (pressureChangeSmallEnough_host == FALSE) {
                    /* redo predictor step with smaller timestep, derivatives are in p_device */
					printf("Reducing timestep due to Pressure Check function to: %.17e\n", dt_host);
                    if (fabs(maxpressureDiff_host -maxpressureDiff_previous) < 1e-3) {
                        maxpressureDiff_cnt++;
                    }
                    maxpressureDiff_previous = maxpressureDiff_host;
                    if (maxpressureDiff_cnt > 1) {
                        printf("Cannot reduce timestep anymore, continuing with dt %.17e and maxpressurediff %.17e", dt_host, maxpressureDiff_host);
                        pressureChangeSmallEnough_host = TRUE;
                    }
				}
                if (pressureChangeSmallEnough_host == TRUE) {
                    /* okay, step seems good, let's do the corrector step */
                    substep_currentTime = currentTime + dt_host;
					currentTime += dt_host;
					printf("Timestep okay, continuing with NBODY integration rk4.\n");
                        // -> rk4 integration with back reaction if wanted
#if GRAVITATING_POINT_MASSES
	                cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
                    rk4_nbodies();
#endif
					cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(currentTimeD), &currentTime, sizeof(double)));
    	            if (param.selfgravity) {
        	            copy_gravitational_accels_device_to_device(&predictor_device, &p_device);
                    }

                    /* now righthandside with predictor variables */
		            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &predictor_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
	                cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
				    rightHandSide();
                    /* now the corrector step  with the original values of p_device and the derivatives of p_device and predictor_device */
		            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &p_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
	                cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
    	            cudaVerifyKernel((CorrectorStep_heun<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>()));
		    //step was successful --> do something (e.g. look for min/max pressure...)
                    afterIntegrationStep();

				}
			}
		} // current time < end time loop
		// write results
#if FRAGMENTATION
        cudaVerify(hipDeviceSynchronize());
        cudaVerifyKernel((damageLimit<<<numberOfMultiprocessors*4, NUM_THREADS_PC_INTEGRATOR>>>()));
        cudaVerify(hipDeviceSynchronize());
#endif
        copyToHostAndWriteToFile(timestep, lastTimestep);

	} // timestep loop

	// free memory

    int free_immutables = 1;
#if GRAVITATING_POINT_MASSES
    for (rkstep = 0; rkstep < 4; rkstep++) {
        free_pointmass_memory(&rk4_pointmass_device[rkstep], free_immutables);
        }
#endif

    free_particles_memory(&predictor_device, free_immutables);
#if GRAVITATING_POINT_MASSES
    free_pointmass_memory(&predictor_pointmass_device, free_immutables);
#endif
	cudaVerify(hipFree(courantPerBlock));
	cudaVerify(hipFree(forcesPerBlock));
    cudaVerify(hipFree(dtSPerBlock));
	cudaVerify(hipFree(dtePerBlock));
	cudaVerify(hipFree(dtrhoPerBlock));
	cudaVerify(hipFree(dtdamagePerBlock));
    cudaVerify(hipFree(dtalphaPerBlock));
    cudaVerify(hipFree(dtbetaPerBlock));
    cudaVerify(hipFree(dtalpha_epsporPerBlock));
    cudaVerify(hipFree(dtepsilon_vPerBlock));
    cudaVerify(hipFree(dtartviscPerBlock));
    cudaVerify(hipFree(maxpressureDiffPerBlock));
}
