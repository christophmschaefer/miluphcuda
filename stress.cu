#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */



#include "stress.h"
#include "parameter.h"
#include "miluph.h"
#include "timeintegration.h"
#include "linalg.h"


#if FRAGMENTATION
// if 1, then damage reduces the principal stresses
// if 0, then p<0 -> (1-d) p and S -> (1-d) S
// disabled for the time being
# define DAMAGE_ACTS_ON_PRINCIPAL_STRESSES 0
#else
# define DAMAGE_ACTS_ON_PRINCIPAL_STRESSES 0
#endif


// principal axes damage does not work for pressure dependent yield strengths
#if DAMAGE_ACTS_ON_PRINCIPAL_STRESSES  &&  ( COLLINS_PLASTICITY || COLLINS_PLASTICITY_SIMPLE )
#error Do not combine DAMAGE_ACTS_ON_PRINCIPAL_STRESSES and COLLINS_PLASTICITY or COLLINS_PLASTICITY_SIMPLE.
#endif


#if SOLID
// here we set the stress tensor sigma from pressure and deviatoric stress S
// note, that S was already lowered in plasticity
__global__ void set_stress_tensor(void)
{
    register int i, inc, matId;
    int d, e;
    int niters;
    double sigma[DIM][DIM];
# if DAMAGE_ACTS_ON_PRINCIPAL_STRESSES
    double sigmatmp[DIM][DIM];
    double rotation_matrix[DIM][DIM];
    double main_stresses[DIM];
# endif
    double damage = 0.0;

    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        niters = 0;

# if FRAGMENTATION
        damage = p.damage_total[i];
        if (damage > 1.0) damage = 1.0;
        if (damage < 0.0) damage = 0.0;
# else
        damage = 0.0;
# endif

# if DAMAGE_ACTS_ON_PRINCIPAL_STRESSES
        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
                sigmatmp[d][e] = 0.0;
                sigma[d][e] = p.S[stressIndex(i, d, e)];
                if (d == e) {
                    sigma[d][e] += -p.p[i];
                }
            }
        }
        // calculate main stresses
        niters = calculate_all_eigenvalues(sigma, main_stresses, rotation_matrix);
        for (d = 0; d < DIM; d++) {
            sigmatmp[d][d] = main_stresses[d];
            if (sigmatmp[d][d] > 0) {
                sigmatmp[d][d] *= (1.0 - damage);
            }
        }
        // rotate back the lowered principal stresses
        multiply_matrix(sigmatmp, rotation_matrix, sigma);
        transpose_matrix(rotation_matrix);
        multiply_matrix(rotation_matrix, sigma, sigmatmp);

        // sigmatmp now holds the stress tensor for particle i with damaged reduced stresses
        copy_matrix(sigmatmp, sigma);
# else
        // assemble stress tensor
        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
#  if COLLINS_PLASTICITY || COLLINS_PLASTICITY_SIMPLE
                // for the Collins model the damage directly affects S via the yield strength, therefore not (additionally) reduced here
                sigma[d][e] = p.S[stressIndex(i, d, e)];
#  else
                // reduction of S following Grady-Kipp model
                sigma[d][e] = (1.0 - damage) * p.S[stressIndex(i, d, e)];
#  endif
                // the pure pressure part of sigma is always reduced for p < 0
                if (d == e) { // the trace
                    if (p.p[i] < 0) {
                        sigma[d][e] += - (1.0 - damage) * p.p[i];
                    } else {
                        sigma[d][e] += -p.p[i];
                    }
                }
            }
        }
# endif

        // remember sigma
        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
                p_rhs.sigma[stressIndex(i,d,e)] = sigma[d][e];
            }
        }

    }
}
#endif  // SOLID
