#include "hip/hip_runtime.h"
/**
 * @author      Oliver Wandel and Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "miluph.h"
#include "timeintegration.h"
#include "porosity.h"
#include "pressure.h"
#include "parameter.h"
#include "math.h"
#include "float.h"

#if PALPHA_POROSITY

__global__ void calculateDistensionChange()
{
/*
    register int i, inc, matId;
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (matEOS[matId] == EOS_TYPE_JUTZI || matEOS[matId] == EOS_TYPE_JUTZI_MURNAGHAN || matEOS[matId] == EOS_TYPE_JUTZI_ANEOS) {
            if (p.alpha_jutzi[i] <= 1.0) {
                p.dalphadt[i] = 0.0;
                p.alpha_jutzi[i] = 1.0;
            } else {
                p.dalphadt[i] = ((p.dedt[i] * p.delpdele[i] + p.alpha_jutzi[i] * p.drhodt[i] * p.delpdelrho[i])
                            * p.dalphadp[i]) / (p.alpha_jutzi[i] + p.dalphadp[i] * (p.p[i] - p.rho[i] * p.delpdelrho[i]));
                if (p.dalphadt[i] > 0.0) {
                    p.dalphadt[i] = 0.0;
                }
	        }
        } else {
            p.dalphadt[i] = 0.0;
        }
    }
*/
}
#endif

#if SIRONO_POROSITY
#define MAXFLOAT DBL_MAX

__global__ void calculateCompressiveStrength()
{
    register int i, inc, matId;
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (matEOS[matId] == EOS_TYPE_SIRONO) {
            double alpha = matporsirono_alpha[matId];
            double pm = matporsirono_pm[matId];
            double phimax = matporsirono_phimax[matId];
            double phi0 = matporsirono_phi0[matId];
            double delta = matporsirono_delta[matId];
            double rho_s = matporsirono_rho_s[matId];
            double phi = p.rho[i] / rho_s;
            /* Using omni-sided_compression curve for compressive strength */
            if (phi <= 0.125)
                p.compressive_strength[i] = alpha * 31.45166;
            if ((phi > 0.125) && (phi < 0.58))
                p.compressive_strength[i] = alpha * pm * pow(((phimax - phi0) / (phimax - phi) - 1.0), delta * 2.302585);
            if (phi >= 0.58)
                p.compressive_strength[i] = MAXFLOAT;
        } else {
            p.compressive_strength[i] = MAXFLOAT;
        }
    }
}

__global__ void calculateTensileStrength()
{
    register int i, inc, matId;
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (matEOS[matId] == EOS_TYPE_SIRONO) {
            double rho_s = matporsirono_rho_s[matId];
            double phi = p.rho[i] / rho_s;
            double tensStrength;
            tensStrength = pow(10.0, (2.8 + 1.48 * phi));
            p.tensile_strength[i] = tensStrength * (-1.0);
        } else {
            p.tensile_strength[i] = -MAXFLOAT;
        }
    }
}

#endif
