#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "density.h"
#include "miluph.h"
#include "config_parameter.h"
#include "timeintegration.h"
#include "parameter.h"
#include "pressure.h"
#include "tree.h"

extern __device__ SPH_kernel kernel;
extern __device__ SPH_kernel wendlandc2_p;
#if SML_CORRECTION
extern __device__ void redo_NeighbourSearch(int particle_id, int *interactions);
#endif // SML_CORRECTION

// calculates the density of all particles via the kernel sum
// is also called for INTEGRATE_DENSITY to determine the densities of particles
// of materials with density_via_kernel_sum = 1 in material.cfg
__global__ void calculateDensity(int *interactions) {
    register int64_t interactions_index;
    int i;
    int j;
    int inc;
    int ip;
    int d;
    double W;
    double Wj;
    double dx[DIM];
    double dWdx[DIM];
    double dWdr;
    double rho;
    double sml;
    double tolerance;
#if SML_CORRECTION
    double dhdrho, sml_omega,sml_omega_sum, r;
    double f, df, h_new, h_init, rho_h;
    //the proportionality constant (h_fact = 4.0) defines the average number of neighbours: [2D] noi = pi * h_fact^2, [3D] noi = 4/3 * pi * h_fact^3
    double h_fact = 4.0;
#endif // SML_CORRECTION
    
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
#if INTEGRATE_DENSITY
        if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[i]] || p_rhs.materialId[i] == EOS_TYPE_IGNORE || matdensity_via_kernel_sum[p_rhs.materialId[i]] < 1) {
                continue;
        }
#else
        if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[i]] || p_rhs.materialId[i] == EOS_TYPE_IGNORE) {
                continue;
        }
#endif // INTEGRATE_DENSITY
        tolerance = 0.0;
        int cnt = 0;
        
#if SML_CORRECTION
        h_init = p.h[i];
        h_new = 0.0;
        /* // if Bisection method is used
        double a = 0.0, b = 0.0, c = 0.0;
	    int bis_cnt = 0;
        int bisection = 0; */
#endif // SML_CORRECTION

        do {
#if SML_CORRECTION
            sml_omega_sum = 0.0;
#endif // SML_CORRECTION
            sml = p.h[i];

            // self density is m_i W_ii
            for (d = 0; d < DIM; d++) {
                dx[d] = 0;
            }
            kernel(&W, dWdx, &dWdr, dx, sml);
#if SHEPARD_CORRECTION
            W /= p_rhs.shepard_correction[i];
#endif
            rho = p.m[i] * W;
            if (rho == 0.0) {
                printf("rho is %f W: %e \n", rho, W);
            }
            // sph sum for particle i
            for (j = 0; j < p.noi[i]; j++) {
                interactions_index = (int64_t)i * MAX_NUM_INTERACTIONS + j;
                ip = interactions[interactions_index];
                if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[ip]] || p_rhs.materialId[ip] == EOS_TYPE_IGNORE) {
                    continue;
                }
#if (VARIABLE_SML || INTEGRATE_SML || DEAL_WITH_TOO_MANY_INTERACTIONS)
                sml = 0.5*(p.h[i] + p.h[ip]);
#endif

                dx[0] = p.x[i] - p.x[ip];
#if DIM > 1
                dx[1] = p.y[i] - p.y[ip];
#if DIM > 2
                dx[2] = p.z[i] - p.z[ip];
#endif
#endif

#if SML_CORRECTION
                r = 0;
                for (d = 0; d < DIM; d++) {
                    r += dx[d]*dx[d];
                }
                r = sqrt(r);
#endif // SML_CORRECTION

#if AVERAGE_KERNELS
                kernel(&W, dWdx, &dWdr, dx, p.h[i]);
                Wj = 0;
                kernel(&Wj, dWdx, &dWdr, dx, p.h[j]);
# if SHEPARD_CORRECTION
                W /= p_rhs.shepard_correction[i];
                Wj /= p_rhs.shepard_correction[j];
# endif
                W = 0.5 * (W + Wj);
#else
                kernel(&W, dWdx, &dWdr, dx, sml);
# if SHEPARD_CORRECTION
                W /= p_rhs.shepard_correction[i];
# endif
            // contribution of interaction
#endif // AVERAGE_KERNELS

#if SML_CORRECTION
                sml_omega_sum += p.m[ip] * (-1) * (DIM * W/sml + (r / sml) * dWdr);
#endif // SML_CORRECTION
                rho += p.m[ip] * W;
            }
#if SML_CORRECTION
            rho_h = p.m[i] * pow(double(h_fact / p.h[i]), DIM);
            dhdrho = -p.h[i] / (DIM * rho);
            sml_omega = 1 - dhdrho * sml_omega_sum;

            // Newton-Raphson method tolerance e-3 (Phantom)
            f = rho_h - rho;
            df = -DIM * rho / p.h[i] * sml_omega;
            h_new = p.h[i] - f / df;

            // arbitrary set limit for sml change
            if (h_new > 1.2 * p.h[i]) {
                h_new = 1.2 * p.h[i];
            } else if (h_new < 0.8 * p.h[i]) {
                h_new = 0.8 * p.h[i];
            }
/*
            //Bisection method (alternative to NR method)
	        if (cnt == 0 && h_new < 0) {
	            bisection = 1;
	        }
	        if (bisection == 1) {
	    	    if ((f/df) > 0) {
		            if(bis_cnt == 0) {
		    	        b = p.h[i];
	                } else {
		    	        b = c;
		            }
	    	    } else if((f/df) < 0) {
                    if(bis_cnt == 0) {
                    	a = p.h[i];
                    	b = 2.0 * a;
                    } else {
                    	a = c;
                    }
		        }
		        c = 0.5 * (a + b);
                h_init = p.h[i];
                h_new = c;
                bis_cnt++;
	        }
*/
           	tolerance = abs(h_new - p.h[i]) / h_init;
            if (tolerance > 1e-3) {
                if (h_new < 0){
	       	        printf("SML_CORRECTION: NEGATIVE SML!");
                }
                p.h[i] = h_new;
                p.sml_omega[i] = sml_omega;
                redo_NeighbourSearch(i, interactions);
                cnt++;
            }
#endif // SML_CORRECTION
        } while (tolerance > 1e-3 && cnt < 10);       
        // write to global memory
        p.rho[i] = rho;
    }
}
