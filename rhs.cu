#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com and Thomas I. Maindl
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "timeintegration.h"
#include "rhs.h"
#include "miluph.h"
#include "parameter.h"
#include "tree.h"
#include "boundary.h"
#include "density.h"
#include "plasticity.h"
#include "porosity.h"
#include "pressure.h"
#include "soundspeed.h"
#include "gravity.h"
#include "xsph.h"
#include "internal_forces.h"
#include "velocity.h"
#include "little_helpers.h"
#include "viscosity.h"
#include "artificial_stress.h"
#include "stress.h"
#include "damage.h"

extern int flag_force_gravity_calc;
extern int gravity_index;
extern __device__ int movingparticles;
extern __device__ int reset_movingparticles;

extern __device__ volatile int maxNodeIndex;

// tree computational domain
extern double *minxPerBlock, *maxxPerBlock;
extern __device__ double minx, maxx;
#if DIM > 1
extern double *minyPerBlock, *maxyPerBlock;
extern __device__ double miny, maxy;
#endif
#if DIM == 3
extern double *minzPerBlock, *maxzPerBlock;
extern __device__ double minz, maxz;
#endif

extern volatile int terminate_flag;

// zero all derivatives
__global__ void zero_all_derivatives(int *interactions)
{
    register int i, inc, dd;
#if SOLID
    register int ddd;
#endif
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        p.ax[i] = 0.0;
#if DIM > 1
        p.ay[i] = 0.0;
#if DIM > 2
        p.az[i] = 0.0;
#endif
#endif
#if INTEGRATE_SML
        p.dhdt[i] = 0.0;
#endif
        p.drhodt[i] = 0.0;
#if INTEGRATE_ENERGY
        p.dedt[i] = 0.0;
#endif
#if SHEPARD_CORRECTION
        p_rhs.shepard_correction[i] = 1.0;
#endif
#if SML_CORRECTION
        p.sml_omega[i] = 1.0;
#endif
#if SOLID
        for (dd = 0; dd < DIM*DIM; dd++) {
            p.dSdt[i*DIM*DIM+dd] = 0.0;
            p_rhs.sigma[i*DIM*DIM+dd] = 0.0;
        }
#if TENSORIAL_CORRECTION
        for (dd = 0; dd < DIM; dd++) {
            for (ddd = 0; ddd < DIM; ddd++) {
                p_rhs.tensorialCorrectionMatrix[i*DIM*DIM+dd*DIM+ddd] = 0.0;
                if (dd == ddd) {
                    p_rhs.tensorialCorrectionMatrix[i*DIM*DIM+dd*DIM+ddd] = 1.0;
                }
            }
        }
#endif
#endif
        // reset all interactions
        for (dd = 0; dd < MAX_NUM_INTERACTIONS; dd++) {
            interactions[i*MAX_NUM_INTERACTIONS + dd] = -1;
        }

#if FRAGMENTATION
        p.dddt[i] = 0.0;
#endif

    }
#if GRAVITATING_POINT_MASSES
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numPointmasses; i += inc) {
        pointmass.ax[i] = 0.0;
        pointmass.feedback_ax[i] = 0.0;
#if DIM > 1
        pointmass.ay[i] = 0.0;
        pointmass.feedback_ay[i] = 0.0;
#if DIM > 2
        pointmass.az[i] = 0.0;
        pointmass.feedback_az[i] = 0.0;
#endif
#endif
    }
#endif // GRAVITATING_POINT_MASSES
}


/* determine all derivatives */
void rightHandSide()
{
#if DEBUG_RHS_RUNTIMES
    hipEvent_t start, stop;
    float time[MAX_NUMBER_PROFILED_KERNELS];
    float totalTime = 0.0;
    int timerCounter = 0;
#endif
#if DEBUG_TREE
    double xmin, xmax, ymin, ymax, zmin, zmax;
    double radiusmax, radiusmin;
    int *treeDepthPerBlock;
    int maxtreedepth_host = 0;
    int maxNodeIndex_host;
#endif
    int *movingparticlesPerBlock;
    int movingparticles_host = 0;
    int calculate_nbody = 0;

#if GRAVITATING_POINT_MASSES
    if (param.integrator_type == HEUN_RK4) {
        calculate_nbody = 0;
    } else {
        calculate_nbody = 1;
    }
#endif

#if USE_SIGNAL_HANDLER
    if (terminate_flag) {
        copyToHostAndWriteToFile(-2, -2);
    }
#endif

#if DEBUG_RHS_RUNTIMES
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    cudaVerify(hipMemset(childListd, EMPTY, memorySizeForChildren));
    cudaVerify(hipDeviceSynchronize());

#if DEBUG_RHS || DEBUG_TIMESTEP
    fprintf(stdout, "rhs call\n");
#endif

    // zero all accelerations
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
    cudaVerifyKernel((zero_all_derivatives<<<numberOfMultiprocessors, NUM_THREADS_256>>>(interactions)));
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration zeroing all: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif

    // check if boundary conditions are violated
    cudaVerifyKernel((BoundaryConditionsBeforeRHS<<<16 * numberOfMultiprocessors, NUM_THREADS_BOUNDARY_CONDITIONS>>>(interactions)));

    cudaVerify(hipDeviceSynchronize());

#if GHOST_BOUNDARIES
    /*
       the location of the ghost boundary particles are set. The quantities for the ghost particles will
       be set later on as soon as we know the quantities for the real particles (density, pressure...)
     */
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((insertGhostParticles<<<4 * numberOfMultiprocessors, NUM_THREADS_BOUNDARY_CONDITIONS>>>()));
    //cudaVerifyKernel((insertGhostParticles<<<1, 1>>>()));
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration inserting ghost particles: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

    cudaVerify(hipDeviceSynchronize());

#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
    cudaVerifyKernel((computationalDomain<<<numberOfMultiprocessors, NUM_THREADS_COMPUTATIONAL_DOMAIN>>>(
                    minxPerBlock, maxxPerBlock
#if DIM > 1
                    , minyPerBlock, maxyPerBlock
#endif
#if DIM == 3
                    , minzPerBlock, maxzPerBlock
#endif
                    )));
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration comp domain: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif

    cudaVerify(hipDeviceSynchronize());

#if DEBUG_TREE
    hipMemcpyFromSymbol(&xmin, HIP_SYMBOL(minx), sizeof(double));
    hipMemcpyFromSymbol(&xmax, HIP_SYMBOL(maxx), sizeof(double));
    radiusmax = xmax - xmin;
# if DIM > 1
    hipMemcpyFromSymbol(&ymin, HIP_SYMBOL(miny), sizeof(double));
    hipMemcpyFromSymbol(&ymax, HIP_SYMBOL(maxy), sizeof(double));
    radiusmax = max(radiusmax, ymax-ymin);
# endif
# if DIM == 3
    hipMemcpyFromSymbol(&zmin, HIP_SYMBOL(minz), sizeof(double));
    hipMemcpyFromSymbol(&zmax, HIP_SYMBOL(maxz), sizeof(double));
    radiusmax = max(radiusmax, zmax-zmin);
# endif
    printf("computational domain: x [%e, %e]", xmin, xmax);
# if DIM > 1
    printf(", y [%e, %e]", ymin, ymax);
# endif
# if DIM == 3
    printf(", z [%e, %e]", zmin, zmax);
# endif
    printf("\n");
#endif  // DEBUG_TREE

#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
    cudaVerifyKernel((buildTree<<<numberOfMultiprocessors, NUM_THREADS_BUILD_TREE>>>()));
    cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration build tree: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif

#if DEBUG_TREE
    hipMemcpyFromSymbol(&maxNodeIndex_host, HIP_SYMBOL(maxNodeIndex), sizeof(int));
    fprintf(stdout, "number of inner nodes: %d\n", (numberOfNodes - maxNodeIndex_host));
    fprintf(stdout, "number of used inner nodes / number of allocated nodes: %.7f %%\n",
            100.0 * (float)(numberOfNodes - maxNodeIndex_host) / (float)(numberOfNodes - numberOfParticles));
    // get maximum depth of tree
    cudaVerify(hipMalloc((void**)&treeDepthPerBlock, sizeof(int)*numberOfMultiprocessors));
    cudaVerifyKernel((getTreeDepth<<<numberOfMultiprocessors, NUM_THREADS_TREEDEPTH>>>(treeDepthPerBlock)));
    hipMemcpyFromSymbol(&maxtreedepth_host, HIP_SYMBOL(treeMaxDepth), sizeof(int));
    fprintf(stdout, "max depth of tree: %d\n", maxtreedepth_host);
    radiusmin = radiusmax * pow(0.5, maxtreedepth_host-1);
    fprintf(stdout, "largest node length: %g \t smallest node length: %g\n", radiusmax, radiusmin);
    cudaVerify(hipFree(treeDepthPerBlock));
#endif

    cudaVerify(hipDeviceSynchronize());

#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
#if VARIABLE_SML
    // boundary conditions for sml
# if DEBUG_RHS
    printf("calling check_sml_boundary\n");
# endif
    cudaVerifyKernel((check_sml_boundary<<<numberOfMultiprocessors * 4, NUM_THREADS_NEIGHBOURSEARCH>>>()));
    cudaVerify(hipDeviceSynchronize());
#endif
#if VARIABLE_SML && FIXED_NOI
    // call only for the fixed number of interactions case
    // if INTEGRATE_SML, the sml is integrated and we only need to symmetrize the interactions later on
# if DEBUG_RHS
    printf("calling knnNeighbourSearch\n");
# endif
    cudaVerifyKernel((knnNeighbourSearch<<<numberOfMultiprocessors * 4, NUM_THREADS_NEIGHBOURSEARCH>>>(
                    interactions)));
    cudaVerify(hipDeviceSynchronize());
#endif
#if DEAL_WITH_TOO_MANY_INTERACTIONS // make sure that a particle does not get more than MAX_NUM_INTERACTIONS
# if DEBUG_RHS
    printf("calling nearNeighbourSearch_modify_sml\n");
# endif
    cudaVerifyKernel((nearNeighbourSearch_modify_sml<<<numberOfMultiprocessors * 4, NUM_THREADS_NEIGHBOURSEARCH>>>(
                    interactions)));
#else // risk a termination if MAX_NUM_INTERACTIONS is reached for one particle
# if DEBUG_RHS
    printf("calling nearNeighbourSearch\n");
# endif
    cudaVerifyKernel((nearNeighbourSearch<<<numberOfMultiprocessors * 4, NUM_THREADS_NEIGHBOURSEARCH>>>(
                    interactions)));
#endif
    cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration neighboursearch: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif
    cudaVerifyKernel((setEmptyMassForInnerNodes<<<numberOfMultiprocessors * 4, NUM_THREADS_512>>>()));
    cudaVerify(hipDeviceSynchronize());
    // TODO: only if debug
#if 0
    hipMemcpy(p_host.noi, p_device.noi, memorySizeForInteractions, hipMemcpyDeviceToHost);
    cudaVerify(hipDeviceSynchronize());
    int i;
    int maxNumInteractions = 0;
    for (i = 0; i < numberOfParticles; i++) {
        maxNumInteractions = max(maxNumInteractions, p_host.noi[i]);
        if (maxNumInteractions > MAX_NUM_INTERACTIONS) {
            fprintf(stderr, "max num interactions exceeded by particle %d\n", i);
            exit(1);
        }
    }
    printf("maximum number of interactions: %d\n", maxNumInteractions);
#endif

//#if !INTEGRATE_DENSITY
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((calculateDensity<<<numberOfMultiprocessors * 4, NUM_THREADS_DENSITY>>>( interactions)));
//    cudaVerifyKernel((calculateDensity<<<1,1>>>( interactions)));
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration density: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
//#endif

#if SHEPARD_CORRECTION
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((shepardCorrection<<<numberOfMultiprocessors*4, NUM_THREADS_256>>>( interactions)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration shepard correction: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
    //cudaVerifyKernel((printTensorialCorrectionMatrix<<<1,1>>>( interactions)));
#endif

#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
    cudaVerifyKernel((calculateSoundSpeed<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration soundspeed: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif

#if (NAVIER_STOKES || BALSARA_SWITCH || INVISCID_SPH)
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((CalcDivvandCurlv<<<numberOfMultiprocessors * 4, NUM_THREADS_128>>>(
                    interactions)));
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration div v and curl v: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if SIRONO_POROSITY
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((calculateCompressiveStrength<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    cudaVerify(hipDeviceSynchronize());
    cudaVerifyKernel((calculateTensileStrength<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration compressive, tensile and shear strength: %.2f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if PURE_REGOLITH
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((plasticity<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration plasticity: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
    cudaVerifyKernel((calculatePressure<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration pressure: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif
/*  function is not in porosity.cu anymore but in timeintecration.cu internal forces
#if PALPHA_POROSITY
    cudaVerifyKernel((calculateDistensionChange<<<numberOfMultiprocessors * 4, NUM_THREADS_PALPHA_POROSITY>>>()));
    cudaVerify(hipDeviceSynchronize());
#endif
*/

    if (param.selfgravity) {
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(start, 0);
#endif
        cudaVerifyKernel((calculateCentersOfMass<<<1, NUM_THREADS_CALC_CENTER_OF_MASS>>>()));
        cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time[timerCounter], start, stop);
        printf("duration calc center of mass: %.7f ms\n", time[timerCounter]);
        totalTime += time[timerCounter++];
#endif
    }

#if INVISCID_SPH
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((betaviscosity<<<numberOfMultiprocessors * 4, NUM_THREADS_128>>>(
		    interactions)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration betaviscosity: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if (SYMMETRIC_STRESSTENSOR || FRAGMENTATION || PLASTICITY)
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((symmetrizeStress<<<4 * numberOfMultiprocessors, NUM_THREADS_512>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration symmetrize stress tensor: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if FRAGMENTATION
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((damageLimit<<<numberOfMultiprocessors*4, NUM_THREADS_512>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration damage limit: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
    fflush(stdout);
#endif

#if PLASTICITY
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((plasticityModel<<<numberOfMultiprocessors * 4, NUM_THREADS_512>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration plasticityModel: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if JC_PLASTICITY
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((JohnsonCookPlasticity<<<numberOfMultiprocessors * 4, NUM_THREADS_512>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration johnson-cook: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if TENSORIAL_CORRECTION
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((tensorialCorrection<<<numberOfMultiprocessors*4, NUM_THREADS_256>>>( interactions)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration tensorial correction: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
//    cudaVerifyKernel((printTensorialCorrectionMatrix<<<1,1>>>( interactions)));
#endif

#if VISCOUS_REGOLITH
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((calculatedeviatoricStress<<<numberOfMultiprocessors*4, NUM_THREADS_256>>>( interactions)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration viscous regolith : %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if XSPH
    cudaVerify(hipDeviceSynchronize());
    cudaVerifyKernel((calculateXSPHchanges<<<4 * numberOfMultiprocessors, NUM_THREADS_512>>>(interactions)));
#endif /*XSPH */

#if GHOST_BOUNDARIES
    /*
       the location of the ghost boundary particles are set. The quantities for the ghost particles will
       be set later on as soon as we know the quantities for the real particles (density, pressure...)
     */
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((setQuantitiesGhostParticles<<<numberOfMultiprocessors, NUM_THREADS_BOUNDARY_CONDITIONS>>>()));
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration quantities ghost particles: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if DEBUG_MISC
    fprintf(stdout, "checking correlation matrix\n");
    fflush(stdout);
    cudaVerifyKernel((checkNaNs<<<numberOfMultiprocessors, NUM_THREADS_128>>>(interactions)));
    cudaVerify(hipDeviceSynchronize());
    fprintf(stdout, "starting internalForces\n");
    fflush(stdout);
#endif

#if SOLID
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((set_stress_tensor<<<numberOfMultiprocessors, NUM_THREADS_256>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration set stress tensor: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if NAVIER_STOKES
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((calculate_kinematic_viscosity<<<numberOfMultiprocessors, NUM_THREADS_256>>>()));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration calculation kinematic viscosity: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if NAVIER_STOKES
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((calculate_shear_stress_tensor<<<numberOfMultiprocessors, NUM_THREADS_256>>>(interactions)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration calculation shear stress tensor: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif


#if ARTIFICIAL_STRESS
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((compute_artificial_stress<<<numberOfMultiprocessors, NUM_THREADS_256>>>(interactions)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration artificial_stress: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

    // the main loop, where all accelerations are calculated
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
#endif
    cudaVerifyKernel((internalForces<<<numberOfMultiprocessors, NUM_THREADS_128>>>(interactions)));
    //cudaVerifyKernel((internalForces<<<1, 1 >>>(interactions)));
    cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration internal forces: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
#endif

#if GRAVITATING_POINT_MASSES
    // interaction with the point masses
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    cudaVerifyKernel((gravitation_from_point_masses<<<numberOfMultiprocessors, NUM_THREADS_128>>>(calculate_nbody)));
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration gravitation from point masses: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
    // back reaction from the disk
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(start, 0);
# endif
    backreaction_from_disk_to_point_masses(calculate_nbody);
    cudaVerify(hipDeviceSynchronize());
# if DEBUG_RHS_RUNTIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time[timerCounter], start, stop);
    printf("duration backreaction from the particles on pointmasses: %.7f ms\n", time[timerCounter]);
    totalTime += time[timerCounter++];
# endif
#endif

#if DEBUG_MISC
    if (param.verbose) fprintf(stdout, "checking for nans after internal_forces\n");
    fflush(stdout);
    cudaVerifyKernel((checkNaNs<<<numberOfMultiprocessors, NUM_THREADS_128>>>(interactions)));
    cudaVerify(hipDeviceSynchronize());
    if (param.verbose) fprintf(stdout, "starting internalForces\n");
    fflush(stdout);
#endif

#if GHOST_BOUNDARIES
    cudaVerifyKernel((removeGhostParticles<<<1,1>>>()));
    cudaVerify(hipDeviceSynchronize());
#endif


    /* check if we need the nbody-tree stuff has to be re-organised or
       if we could use the node masses and positions of last time step */

    if (param.selfgravity && param.decouplegravity) {
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(start, 0);
#endif
        if (gravity_index%10 == 0) {
            flag_force_gravity_calc = 1;
        }
        /* alloc mem */
        cudaVerify(hipMalloc((void**)&movingparticlesPerBlock, sizeof(int)*numberOfMultiprocessors));
        /* determine how many particles will change their node */
        cudaVerifyKernel(((measureTreeChange<<<numberOfMultiprocessors, NUM_THREADS_TREECHANGE>>>(movingparticlesPerBlock))));
        /* get number of changing particles */
        hipMemcpyFromSymbol(&movingparticles_host, HIP_SYMBOL(movingparticles), sizeof(int));
        double changefraction = movingparticles_host*1.0/numberOfParticles;
#if DEBUG_GRAVITY
        fprintf(stdout, "%d particles change their nodes, this is a fraction of %g %% (currently allowed max is 0.1 %%)\n",
                movingparticles_host, changefraction*1e2);
#endif
        if (changefraction > 1e-3) {
            flag_force_gravity_calc = 1;
            hipMemcpyToSymbol(HIP_SYMBOL(reset_movingparticles), &flag_force_gravity_calc, sizeof(int));
        }
        /* free mem */
        cudaVerify(hipFree(movingparticlesPerBlock));
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time[timerCounter], start, stop);
        printf("duration tree changes: %.7f ms\n", time[timerCounter]);
        totalTime += time[timerCounter++];
#endif
    }

    /* self-gravitation using TREE */
    if (param.selfgravity) {
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(start, 0);
#endif
        if (!param.decouplegravity)
            flag_force_gravity_calc = 1;
        if (flag_force_gravity_calc) {
#if DEBUG_GRAVITY
            fprintf(stdout, "calculating self-gravity using new tree\n");
#endif
            cudaVerifyKernel((selfgravity<<<16*numberOfMultiprocessors, NUM_THREADS_SELFGRAVITY>>>()));
            flag_force_gravity_calc = 0;
            hipMemcpyToSymbol(HIP_SYMBOL(reset_movingparticles), &flag_force_gravity_calc, sizeof(int));
        } else {
#if DEBUG_GRAVITY
            printf("skipping calculation of self-gravity, using values from last timestep\n");
#endif
            cudaVerifyKernel((addoldselfgravity<<<16*numberOfMultiprocessors, NUM_THREADS_SELFGRAVITY>>>()));
        }
        cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time[timerCounter], start, stop);
        printf("duration selfgravity: %.7f ms\n", time[timerCounter]);
        totalTime += time[timerCounter++];
#endif
        gravity_index++;
    }

    /* self gravitation using particle-particle forces */
    if (param.directselfgravity) {
#if DEBUG_GRAVITY
        fprintf(stdout, "calculating self-gravity using n**2 algorithm\n");
#endif
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(start, 0);
#endif
        cudaVerifyKernel((direct_selfgravity<<<numberOfMultiprocessors, NUM_THREADS_SELFGRAVITY>>>()));
        cudaVerify(hipDeviceSynchronize());
#if DEBUG_RHS_RUNTIMES
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time[timerCounter], start, stop);
        printf("duration selfgravity: %.7f ms\n", time[timerCounter]);
        totalTime += time[timerCounter++];
#endif
    }


    /* set any special particle values */
    cudaVerifyKernel((BoundaryConditionsAfterRHS<<<16 * numberOfMultiprocessors, NUM_THREADS_BOUNDARY_CONDITIONS>>>(interactions)));

    // set dx/dt = v or dx/dt = v + dxsph/dt
    cudaVerifyKernel((setlocationchanges<<<4 * numberOfMultiprocessors, NUM_THREADS_512>>>(interactions)));


#if 0 // disabled, cms 2019-12-03: should be sufficient to do this at start of rhs
#if VARIABLE_SML && !READ_INITIAL_SML_FROM_PARTICLE_FILE
    // boundary conditions for the smoothing lengths
# if DEBUG_RHS
    printf("calling check_sml_boundary\n");
# endif
    cudaVerifyKernel((check_sml_boundary<<<numberOfMultiprocessors * 4, NUM_THREADS_NEIGHBOURSEARCH>>>()));
    cudaVerify(hipDeviceSynchronize());
#endif
#endif // 0

#if DEBUG_RHS_RUNTIMES
    fprintf(stdout, "total duration rhs: %.7f ms\n", totalTime);
    if (param.performanceTest)
        write_performance(time);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif
}
