#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */



#include "predictor_corrector_euler.h"
#include "timeintegration.h"
#include "config_parameter.h"
#include "parameter.h"
#include "memory_handling.h"
#include "miluph.h"
#include "pressure.h"
#include "rhs.h"
#include "damage.h"
#include <float.h>

/* predictor corrector scheme with an initial step of dt/2 and the corrector step with dt */


extern __device__ double endTimeD, currentTimeD;
extern __device__ double substep_currentTimeD;
extern __device__ double dt;
extern __device__ double dtmax;
extern __device__ int blockCount;
extern __device__ double emin_d;
extern __device__ double Smin_d;
extern __device__ double rhomin_d;
extern __device__ double damagemin_d;
extern __device__ double alphamin_d;
extern __device__ double betamin_d;
extern __device__ double alpha_epspormin_d;
extern __device__ double epsilon_vmin_d;
extern __device__ int pressureChangeSmallEnough;
extern __device__ double maxpressureDiff;


extern double L_ini;


__global__ void CorrectorStep()
{
    register int i;
#if SOLID
    register int j;
    register int k;
#endif

#if GRAVITATING_POINT_MASSES
    // pointmass loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numPointmasses; i+= blockDim.x * gridDim.x) {
        pointmass.x[i] = pointmass.x[i] + dt * predictor_pointmass.vx[i];
#if DIM > 1
        pointmass.y[i] = pointmass.y[i] + dt * predictor_pointmass.vy[i];
        pointmass.vy[i] = pointmass.vy[i] + dt * predictor_pointmass.ay[i];
        pointmass.ay[i] = predictor_pointmass.ay[i];
#endif
        pointmass.vx[i] = pointmass.vx[i] + dt * predictor_pointmass.ax[i];
        pointmass.ax[i] = predictor_pointmass.ax[i];
#if DIM == 3
        pointmass.z[i] = pointmass.z[i] + dt * predictor_pointmass.vz[i];
        pointmass.vz[i] = pointmass.vz[i] + dt * predictor_pointmass.az[i];
        pointmass.az[i] = predictor_pointmass.az[i];
#endif
    }
#endif // GRAVITATING_POINT_MASSES

    // particle loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        p.x[i] = p.x[i] + dt * predictor.dxdt[i];
#if DIM > 1
        p.y[i] = p.y[i] + dt * predictor.dydt[i];
        p.vy[i] = p.vy[i] + dt * predictor.ay[i];
        p.ay[i] = predictor.ay[i];
#endif
        p.vx[i] = p.vx[i] + dt * predictor.ax[i];
        p.ax[i] = predictor.ax[i];
#if DIM == 3
        p.z[i] = p.z[i] + dt * predictor.dzdt[i];
        p.vz[i] = p.vz[i] + dt * predictor.az[i];
        p.az[i] = predictor.az[i];
#endif
#if INTEGRATE_DENSITY
        p.rho[i] = p.rho[i] + dt * predictor.drhodt[i];
        p.drhodt[i] = predictor.drhodt[i];
#else
        p.rho[i] = p.rho[i];
#endif
#if INTEGRATE_ENERGY
        p.e[i] = p.e[i] + dt * predictor.dedt[i];
        p.dedt[i] = predictor.dedt[i];
#endif
#if FRAGMENTATION
        p.d[i] = p.d[i] + dt * predictor.dddt[i];
        p.dddt[i] = predictor.dddt[i];
#endif
#if INTEGRATE_SML
        p.h[i] = p.h[i] + dt * predictor.dhdt[i];
        p.dhdt[i] = predictor.dhdt[i];
#else
        p.h[i] = predictor.h[i];
#endif
#if JC_PLASTICITY
        p.ep[i] = p.ep[i] + dt * predictor.edotp[i];
        p.edotp[i] = predictor.edotp[i];
        p.T[i] = p.T[i] + dt * predictor.dTdt[i];
        p.dTdt[i] = predictor.dTdt[i];
#endif
#if SIRONO_POROSITY
        p.rho_0prime[i] = p.rho_0prime[i];
        p.rho_c_plus[i] = p.rho_c_plus[i];
        p.rho_c_minus[i] = p.rho_c_minus[i];
        p.compressive_strength[i] = p.compressive_strength[i];
        p.tensile_strength[i] = p.tensile_strength[i];
        p.shear_strength[i] = p.shear_strength[i];
        p.K[i] = p.K[i];
        p.flag_rho_0prime[i] = p.flag_rho_0prime[i];
        p.flag_plastic[i] = p.flag_plastic[i];
#endif
#if EPSALPHA_POROSITY
        p.alpha_epspor[i] = p.alpha_epspor[i] + dt * predictor.dalpha_epspordt[i];
        p.epsilon_v[i] = p.epsilon_v[i] + dt * predictor.depsilon_vdt[i];
        p.dalpha_epspordt[i] = predictor.dalpha_epspordt[i];
        p.depsilon_vdt[i] = predictor.depsilon_vdt[i];
#endif
#if INVISCID_SPH
        p.beta[i] = p.beta[i] + dt * predictor.dbetadt[i];
        p.dbetadt[i] = predictor.dbetadt[i];
#endif
#if SOLID
        for (j = 0; j < DIM; j++) {
            for (k = 0; k < DIM; k++) {
                p.S[stressIndex(i,j,k)] = p.S[stressIndex(i,j,k)] + dt  * predictor.dSdt[stressIndex(i,j,k)];
                p.dSdt[stressIndex(i,j,k)] = predictor.dSdt[stressIndex(i,j,k)];
            }
        }
#endif
    }
}

__global__ void PredictorStep()
{
    register int i;
#if SOLID
    register int j;
    register int k;
#endif


#if GRAVITATING_POINT_MASSES
    // pointmass loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numPointmasses; i+= blockDim.x * gridDim.x) {
        predictor_pointmass.x[i] = pointmass.x[i] + dt/2 * pointmass.vx[i];
        predictor_pointmass.vx[i] = pointmass.vx[i] + dt/2 * pointmass.ax[i];

#if DIM > 1
        predictor_pointmass.y[i] = pointmass.y[i] + dt/2 * pointmass.vy[i];
        predictor_pointmass.vy[i] = pointmass.vy[i] + dt/2 * pointmass.ay[i];
#endif
#if DIM > 2
        predictor_pointmass.z[i] = pointmass.z[i] + dt/2 * pointmass.vz[i];
        predictor_pointmass.vz[i] = pointmass.vz[i] + dt/2 * pointmass.az[i];
#endif
    }
#endif // GRAVITATING_POINT_MASSES


    // particle loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        predictor.x[i] = p.x[i] + dt/2 * p.dxdt[i];
        predictor.vx[i] = p.vx[i] + dt/2 * p.ax[i];

#if DIM > 1
        predictor.y[i] = p.y[i] + dt/2 * p.dydt[i];
        predictor.vy[i] = p.vy[i] + dt/2 * p.ay[i];
#endif
#if DIM > 2
        predictor.z[i] = p.z[i] + dt/2 * p.dzdt[i];
        predictor.vz[i] = p.vz[i] + dt/2 * p.az[i];
#endif
#if INTEGRATE_DENSITY
        predictor.rho[i] = p.rho[i] + dt/2 * p.drhodt[i];
#else
        predictor.rho[i] = p.rho[i];
#endif
#if INTEGRATE_ENERGY
        predictor.e[i] = p.e[i] + dt/2 * p.dedt[i];
#endif
#if INTEGRATE_SML
        predictor.h[i] = p.h[i] + dt/2 * p.dhdt[i];
#else
        predictor.h[i] = p.h[i];
#endif

#if PALPHA_POROSITY
        // p points to p_device and p.p is the pressure at the start of the timestep
        // while predictor changes during the adaptive time step
        predictor.pold[i] = p.p[i];
        predictor.alpha_jutzi[i] = p.alpha_jutzi[i] + dt/2 * p.dalphadt[i];
        predictor.alpha_jutzi_old[i] = p.alpha_jutzi_old[i];
#endif

#if FRAGMENTATION
        predictor.d[i] = p.d[i] + dt/2 * p.dddt[i];
        predictor.numActiveFlaws[i] = p.numActiveFlaws[i];
#if PALPHA_POROSITY
        predictor.damage_porjutzi[i] = p.damage_porjutzi[i] + dt/2 * p.ddamage_porjutzidt[i];
#endif
#endif
#if JC_PLASTICITY
        predictor.ep[i] = p.ep[i] + dt/2 * p.edotp[i];
        predictor.T[i] = p.T[i] + dt/2 * p.dTdt[i];
#endif
#if SIRONO_POROSITY
        predictor.rho_0prime[i] = p.rho_0prime[i];
        predictor.rho_c_plus[i] = p.rho_c_plus[i];
        predictor.rho_c_minus[i] = p.rho_c_minus[i];
        predictor.compressive_strength[i] = p.compressive_strength[i];
        predictor.tensile_strength[i] = p.tensile_strength[i];
        predictor.shear_strength[i] = p.shear_strength[i];
        predictor.K[i] = p.K[i];
        predictor.flag_rho_0prime[i] = p.flag_rho_0prime[i];
        predictor.flag_plastic[i] = p.flag_plastic[i];
#endif
#if EPSALPHA_POROSITY
        predictor.alpha_epspor[i] = p.alpha_epspor[i] + dt/2 * p.dalpha_epspordt[i];
        predictor.epsilon_v[i] = p.epsilon_v[i] + dt/2 * p.depsilon_vdt[i];
#endif
#if INVISCID_SPH
        predictor.beta[i] = p.beta[i] + dt/2 * p.dbetadt[i];
#endif
#if SOLID
        for (j = 0; j < DIM; j++) {
            for (k = 0; k < DIM; k++) {
                predictor.S[stressIndex(i,j,k)] = p.S[stressIndex(i,j,k)] + dt/2 * p.dSdt[stressIndex(i,j,k)];
            }
        }
#endif
    }

}

#if PALPHA_POROSITY
__global__ void CorrectorStepPorous()
{
    register int i;
#if SOLID
    register int j;
    register int k;
#endif

#if GRAVITATING_POINT_MASSES
    // pointmass loop
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numPointmasses; i+= blockDim.x * gridDim.x) {
        predictor_pointmass.x[i] = pointmass.x[i] + dt * predictor_pointmass.vx[i];
#if DIM > 1
        predictor_pointmass.y[i] = pointmass.y[i] + dt * predictor_pointmass.vy[i];
        predictor_pointmass.vy[i] = pointmass.vy[i] + dt * predictor_pointmass.ay[i];
        predictor_pointmass.ay[i] = predictor_pointmass.ay[i];
#endif
        predictor_pointmass.vx[i] = pointmass.vx[i] + dt * predictor_pointmass.ax[i];
        predictor_pointmass.ax[i] = predictor_pointmass.ax[i];
#if DIM == 3
        predictor_pointmass.z[i] = pointmass.z[i] + dt * predictor_pointmass.vz[i];
        predictor_pointmass.vz[i] = pointmass.vz[i] + dt * predictor_pointmass.az[i];
        predictor_pointmass.az[i] = predictor_pointmass.az[i];
#endif
    }
#endif // GRAVITATING_POINT_MASSES


    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        predictor.x[i] = p.x[i] + dt * predictor.dxdt[i];
        predictor.y[i] = p.y[i] + dt * predictor.dydt[i];
        predictor.vx[i] = p.vx[i] + dt * predictor.ax[i];
        predictor.vy[i] = p.vy[i] + dt * predictor.ay[i];
#if DIM == 3
        predictor.z[i] = p.z[i] + dt * predictor.dzdt[i];
        predictor.vz[i] = p.vz[i] + dt * predictor.az[i];
#endif
#if INTEGRATE_DENSITY
        predictor.rho[i] = p.rho[i] + dt * predictor.drhodt[i];
#else
        predictor.rho[i] = p.rho[i];
#endif

#if INTEGRATE_SML
        predictor.h[i] = p.h[i] + dt * predictor.dhdt[i];
#else
        predictor.h[i] = p.h[i];
#endif
#if INTEGRATE_ENERGY
        predictor.e[i] = p.e[i] + dt * predictor.dedt[i];
#endif
#if FRAGMENTATION
        predictor.d[i] = p.d[i] + dt * predictor.dddt[i];
#if PALPHA_POROSITY
        if (predictor.p[i] > p.p[i]) {
            predictor.damage_porjutzi[i] = p.damage_porjutzi[i] + dt * predictor.ddamage_porjutzidt[i];
        } else {
            predictor.d[i] = predictor.d[i];
            predictor.damage_porjutzi[i] = p.damage_porjutzi[i];
        }
#endif
#endif
#if INVISCID_SPH
        predictor.beta[i] = p.beta[i] + dt * predictor.dbetadt[i];
#endif
#if SOLID
        for (j = 0; j < DIM; j++) {
            for (k = 0; k < DIM; k++) {
                predictor.S[stressIndex(i,j,k)] = p.S[stressIndex(i,j,k)] + dt  * predictor.dSdt[stressIndex(i,j,k)];
            }
        }
#endif
#if PALPHA_POROSITY
        /* check if we have compaction and change alpha accordingly */
        if (predictor.p[i] > p.p[i]) {
            predictor.alpha_jutzi[i] = p.alpha_jutzi[i] + dt * predictor.dalphadt[i];
        } else {
            predictor.alpha_jutzi[i] = p.alpha_jutzi[i];
        }
        predictor.alpha_jutzi_old[i] = p.alpha_jutzi_old[i];
#endif
    }
}

/* check the pressure change to avoid large deviation from the crush-curve */
__global__ void pressureChangeCheck(double *maxpressureDiffPerBlock)
{
    __shared__ double sharedMaxpressureDiff[NUM_THREADS_PC_INTEGRATOR];
    double localMaxpressureDiff = 0.0;
    double tmp = 0.0;
    int i, j, k, m;
    maxpressureDiff = 0.0;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        if (matEOS[p_rhs.materialId[i]] == EOS_TYPE_JUTZI || matEOS[p_rhs.materialId[i]] == EOS_TYPE_JUTZI_MURNAGHAN || matEOS[p_rhs.materialId[i]] == EOS_TYPE_JUTZI_ANEOS) {
            // cms - 20190626
            // first rhs is called at beginning of timestep with predictor
            // and at the end with p_device
            tmp = (p.p[i] - predictor.pold[i]);
            localMaxpressureDiff = max(tmp, localMaxpressureDiff);
        }
    }
    i = threadIdx.x;
    sharedMaxpressureDiff[i] = localMaxpressureDiff;
    for (j = NUM_THREADS_PC_INTEGRATOR / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sharedMaxpressureDiff[i] = localMaxpressureDiff = max(localMaxpressureDiff, sharedMaxpressureDiff[k]);
        }
    }
    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        maxpressureDiffPerBlock[k] = localMaxpressureDiff;
        m = gridDim.x - 1;
        if (m == atomicInc((unsigned int *)&blockCount, m)) {
            // last block, so combine all block results
            for (j = 0; j <= m; j++) {
                localMaxpressureDiff = max(localMaxpressureDiff, maxpressureDiffPerBlock[j]);
            }
            maxpressureDiff = localMaxpressureDiff;
            // reset block count
            blockCount = 0;
        }
        if (maxpressureDiff > max_abs_pressure_change) {
            printf("maxpressure change %e\n", maxpressureDiff);
            pressureChangeSmallEnough = FALSE;
            dt = 0.25 * dt;
            dt = min(dt, endTimeD - currentTimeD);
        } else {
            pressureChangeSmallEnough = TRUE;
    //        currentTimeD += dt;
        }
    }
}
#endif

__global__ void setTimestep(double *forcesPerBlock, double *courantPerBlock, double *dtSPerBlock, double *dtePerBlock, double *dtrhoPerBlock, double *dtdamagePerBlock, double *dtalphaPerBlock, double *dtartviscPerBlock, double *dtbetaPerBlock, double *dtalpha_epsporPerBlock, double *dtepsilon_vPerBlock)
{

#define SAFETY_FIRST 0.7

    __shared__ double sharedForces[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedCourant[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedArtVisc[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedS[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharede[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedrho[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double shareddamage[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedalpha[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedbeta[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedalpha_epspor[NUM_THREADS_LIMITTIMESTEP];
    __shared__ double sharedepsilon_v[NUM_THREADS_LIMITTIMESTEP];

    int i, j, k, m;
    int d, dd;
    int index;
    int hasEnergy;
    double forces = DBL_MAX, courant = DBL_MAX;
    double dtx = DBL_MAX;
    double dtS = DBL_MAX;
    double dtrho = DBL_MAX;
    double dte = DBL_MAX;
    double dtdamage = DBL_MAX;
    double dtalpha = DBL_MAX;
    double dtbeta = DBL_MAX;
    double dtalpha_epspor = DBL_MAX;
    double dtepsilon_v = DBL_MAX;
    double temp;
    double sml;
    int matId;
#if SOLID
    double myS, dS;
#endif
    double ax, ay;
#if DIM == 3
    double az;
#endif
    double dtartvisc = DBL_MAX;

    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {
        matId = p_rhs.materialId[i];
#if INTEGRATE_ENERGY
        hasEnergy = 0;

        switch  (matEOS[matId]) {
            case (EOS_TYPE_TILLOTSON):
                hasEnergy = 1;
                break;
            case (EOS_TYPE_JUTZI):
                hasEnergy = 1;
                break;
			case (EOS_TYPE_JUTZI_ANEOS):
				hasEnergy = 1;
				break;
            case (EOS_TYPE_SIRONO):
                hasEnergy = 1;
                break;
            case (EOS_TYPE_EPSILON):
                hasEnergy = 1;
                break;
            case (EOS_TYPE_ANEOS):
                hasEnergy = 1;
                break;
            default:
                hasEnergy = 0;
                break;
        }
#endif
        ax = p.ax[i];
#if DIM > 1
        ay = p.ay[i];
#endif
#if DIM == 3
        az = p.az[i];
#endif
        temp = ax*ax;
#if DIM > 1
        temp += + ay*ay;
#endif
#if DIM == 3
        temp += az*az;
#endif

        sml = p.h[i];
        temp = sqrt(sml / sqrt(temp));
        forces = min(forces, temp);
        temp = sml / p.cs[i];
        courant = min(courant, temp);

#if ARTIFICIAL_VISCOSITY
        temp = COURANT_FACT * sml / (p.cs[i] + 1.2 * (matAlpha[matId]) * p.cs[i] + matBeta[matId] * p.muijmax[i]);
        dtartvisc = min(dtartvisc, temp);
#endif
#if INVISCID_SPH
        if (p.dbetadt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.beta[i])+betamin_d)/fabs(p.dbetadt[i]);
            dtbeta = min(temp, dtbeta);
        }
#endif
#if SOLID
        myS = 0;
        dS = 0;

        for (d = 0; d < DIM; d++) {
            for (dd = 0; dd < DIM; dd++) {
                index = i*DIM*DIM+d*DIM+dd;
                myS += p.S[index]*p.S[index];
                dS += p.dSdt[index]*p.dSdt[index];
            }
        }
        if (dS != 0) {
            temp = SAFETY_FIRST * sqrt((myS+Smin_d)/dS);
            dtS = min(temp, dtS);
        }
#endif
#if INTEGRATE_DENSITY
        if (p.drhodt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.rho[i])+rhomin_d)/fabs(p.drhodt[i]);
            dtrho = min(temp, dtrho);
        }
#endif
#if INTEGRATE_ENERGY
        if (p.dedt[i] != 0 && hasEnergy) {
            temp = SAFETY_FIRST * (fabs(p.e[i])+emin_d)/fabs(p.dedt[i]);
            dte = min(temp, dte);
        }
#endif

#if PALPHA_POROSITY
        if (p.dalphadt[i] != 0) {
            temp = 1.0e-2 / fabs(p.dalphadt[i]);
            dtalpha = min(temp, dtalpha);
        }
#endif

#if EPSALPHA_POROSITY
        if (p.dalpha_epspordt[i] != 0) {
            temp = 1.0e-1 / fabs(p.dalpha_epspordt[i]);
            dtalpha_epspor = min(temp, dtalpha_epspor);
        }

        if (p.depsilon_vdt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.epsilon_v[i])+epsilon_vmin_d)/fabs(p.depsilon_vdt[i]);
            dtepsilon_v = min(temp, dtepsilon_v);
        }
#endif

#if FRAGMENTATION
        if (p.dddt[i] != 0) {
            temp = SAFETY_FIRST * (fabs(p.d[i])+damagemin_d)/fabs(p.dddt[i]);
            dtdamage = min(temp, dtdamage);
        }
#endif


    }
    i = threadIdx.x;
    sharedForces[i] = forces;
    sharedCourant[i] = courant;
    sharedS[i] = dtS;
    sharede[i] = dte;
    sharedrho[i] = dtrho;
    shareddamage[i] = dtdamage;
    sharedalpha[i] = dtalpha;
    sharedbeta[i] = dtbeta;
    sharedalpha_epspor[i] = dtalpha_epspor;
    sharedepsilon_v[i] = dtepsilon_v;

#if ARTIFICIAL_VISCOSITY
    sharedArtVisc[i] = dtartvisc;
#endif
    for (j = NUM_THREADS_LIMITTIMESTEP / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sharedForces[i] = forces = min(forces, sharedForces[k]);
            sharedCourant[i] = courant = min(courant, sharedCourant[k]);
            sharedS[i] = dtS = min(dtS, sharedS[k]);
            sharede[i] = dte = min(dte, sharede[k]);
            sharedrho[i] = dtrho = min(dtrho, sharedrho[k]);
            shareddamage[i] = dtdamage = min(dtdamage, shareddamage[k]);
            sharedalpha[i] = dtalpha = min(dtalpha, sharedalpha[k]);
            sharedalpha_epspor[i] = dtalpha_epspor = min(dtalpha_epspor, sharedalpha_epspor[k]);
            sharedepsilon_v[i] = dtepsilon_v = min(dtepsilon_v, sharedepsilon_v[k]);
#if ARTIFICIAL_VISCOSITY
            sharedArtVisc[i] = dtartvisc = min(dtartvisc, sharedArtVisc[k]);
#endif
#if INVISCID_SPH
            sharedbeta[i] = dtbeta = min(dtbeta, sharedbeta[i]);
#endif
        }
    }
    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        forcesPerBlock[k] = forces;
        courantPerBlock[k] = courant;
        dtSPerBlock[k] = dtS;
        dtePerBlock[k] = dte;
        dtrhoPerBlock[k] = dtrho;
        dtdamagePerBlock[k] = dtdamage;
        dtalphaPerBlock[k] = dtalpha;
        dtalpha_epsporPerBlock[k] = dtalpha_epspor;
        dtepsilon_vPerBlock[k] = dtepsilon_v;
#if ARTIFICIAL_VISCOSITY
        dtartviscPerBlock[k] = dtartvisc;
#endif
#if INVISCID_SPH
        dtbetaPerBlock[k] = dtbeta;
#endif
        m = gridDim.x - 1;
        if (m == atomicInc((unsigned int *)&blockCount, m)) {
            // last block, so combine all block results
            for (j = 0; j <= m; j++) {
                forces = min(forces, forcesPerBlock[j]);
                courant = min(courant, courantPerBlock[j]);
                dtS = min(dtS, dtSPerBlock[j]);
                dte = min(dte, dtePerBlock[j]);
                dtrho = min(dtrho, dtrhoPerBlock[j]);
                dtdamage = min(dtdamage, dtdamagePerBlock[j]);
                dtalpha = min(dtalpha, dtalphaPerBlock[j]);
                dtalpha_epspor = min(dtalpha_epspor, dtalpha_epsporPerBlock[j]);
                dtepsilon_v = min(dtepsilon_v, dtepsilon_vPerBlock[j]);
#if ARTIFICIAL_VISCOSITY
                dtartvisc = min(dtartvisc, dtartviscPerBlock[j]);
#endif
#if INVISCID_SPH
                dtbeta = min(dtbeta, dtbetaPerBlock[j]);
#endif
            }
            // set new timestep
            dt = dtx = min(COURANT_FACT*courant, FORCES_FACT*forces);
#if SOLID
            dt = min(dt, dtS);
#endif
#if INTEGRATE_ENERGY
            dt = min(dt, dte);
#endif
#if INTEGRATE_DENSITY
            dt = min(dt, dtrho);
#endif
#if FRAGMENTATION
            dt = min(dt, dtdamage);
#endif
#if PALPHA_POROSITY
            dt = min(dt, dtalpha);
#endif
#if EPSALPHA_POROSITY
            dt = min(dt, dtalpha_epspor);
            dt = min(dt, dtepsilon_v);
#endif
#if ARTIFICIAL_VISCOSITY
            dt = min(dt, dtartvisc);
#endif
#if INVISCID_SPH
            dt = min(dt, dtbeta);
#endif
            dt = min(dt, endTimeD - currentTimeD);
            if (dt > dtmax) {
                dt = dtmax;
            }
            printf("Time Step Information: dt(v and x): %e dtS: %e dte: %e dtrho: %e dtdamage: %e dtalpha: %e dtalpha_epspor: %e dtepsilon_v: %e\n",
                    dtx, dtS, dte, dtrho, dtdamage, dtalpha, dtalpha_epspor, dtepsilon_v);
            printf("time: %e timestep set to %e, integrating until %e \n", currentTimeD, dt, endTimeD);
#if !PALPHA_POROSITY
            currentTimeD += dt;
#endif
			// reset block count
			blockCount = 0;
		}
	}
}




void predictor_corrector()
{

    double *courantPerBlock, *forcesPerBlock;
    double *dtSPerBlock, *dtePerBlock, *dtrhoPerBlock;
    double *dtdamagePerBlock;
    double *dtalphaPerBlock;
    double *dtartviscPerBlock;
    double *maxpressureDiffPerBlock;
    double *dtbetaPerBlock;
    double *dtalpha_epsporPerBlock;
    double *dtepsilon_vPerBlock;
    int pressureChangeSmallEnough_host;
    double maxpressureDiff_host;
    double maxpressureDiff_previous;
    int maxpressureDiff_cnt;


    cudaVerify(hipMalloc((void**)&courantPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&forcesPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtSPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtePerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtrhoPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtdamagePerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtalphaPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtbetaPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&maxpressureDiffPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtartviscPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtalpha_epsporPerBlock, sizeof(double)*numberOfMultiprocessors));
    cudaVerify(hipMalloc((void**)&dtepsilon_vPerBlock, sizeof(double)*numberOfMultiprocessors));

    int lastTimestep = startTimestep + numberOfTimesteps;
    int timestep;
    double substep_currentTime;
    currentTime = startTime;
    double endTime = startTime;

    int allocate_immutables = 1;
    // alloc mem for one rhs
    allocate_particles_memory(&predictor_device, allocate_immutables);
    copy_particles_immutables_device_to_device(&predictor_device, &p_device);
    copy_particles_variables_device_to_device(&predictor_device, &p_device);
    /* tell the gpu the current time */
    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(currentTimeD), &currentTime, sizeof(double)));
    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(predictor), &predictor_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
    allocate_pointmass_memory(&predictor_pointmass_device, allocate_immutables);
    copy_pointmass_immutables_device_to_device(&predictor_pointmass_device, &pointmass_device);
    /* tell the gpu the current time */
    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(predictor_pointmass), &predictor_pointmass_device, sizeof(struct Pointmass)));
#endif


    for (timestep = startTimestep; timestep < lastTimestep; timestep++) {
        fprintf(stdout, "calculating step %d\n", timestep);
        printf("\nstep %d / %d\n", timestep, lastTimestep);
        endTime += timePerStep;
        fprintf(stdout, " currenttime: %e \t endtime: %e\n", currentTime, endTime);

        /* tell the gpu the time step */
        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(dt), &timePerStep, sizeof(double)));
        /* tell the gpu the end time */
        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(endTimeD), &endTime, sizeof(double)));


        // checking for changes in angular momentum
        if (param.angular_momentum_check > 0) {
            double L_current = calculate_angular_momentum();
            double L_change_relative;
            if (L_ini > 0) {
                L_change_relative = fabs((L_ini - L_current)/L_ini);
            }
            if (param.verbose) {
                fprintf(stdout, "Checking angular momentum conservation.\n");
                fprintf(stdout, "Initial angular momentum: %.17e\n", L_ini);
                fprintf(stdout, "Current angular momentum: %.17e\n", L_current);
                fprintf(stdout, "Relative change: %.17e\n", L_change_relative);
            }
            if (L_change_relative > param.angular_momentum_check) {
                fprintf(stderr, "Conservation of angular momentum violated. Exiting.\n");
                exit(111);
            }
        }





		while (currentTime < endTime) {
			cudaVerify(hipDeviceSynchronize());
			// calculate first right hand side with p_device
	        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &p_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
	        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
            cudaVerify(hipDeviceSynchronize());
            cudaVerify(hipMemcpyFromSymbol(&currentTime, HIP_SYMBOL(currentTimeD), sizeof(double)));
            substep_currentTime = currentTime;
            cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(substep_currentTimeD), &substep_currentTime, sizeof(double)));
            rightHandSide();
            cudaVerify(hipDeviceSynchronize());
            cudaVerifyKernel((setTimestep<<<numberOfMultiprocessors, NUM_THREADS_LIMITTIMESTEP>>>(
                              forcesPerBlock, courantPerBlock,
                              dtSPerBlock, dtePerBlock, dtrhoPerBlock, dtdamagePerBlock,
                              dtalphaPerBlock, dtartviscPerBlock, dtbetaPerBlock, dtalpha_epsporPerBlock, dtepsilon_vPerBlock)));
            cudaVerify(hipDeviceSynchronize());
            /* get the time and the time step from the gpu */
            cudaVerify(hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double)));
            substep_currentTime = currentTime + dt_host * 0.5;

			cudaVerify(hipDeviceSynchronize());
#if PALPHA_POROSITY
            maxpressureDiff_cnt = 0;
            maxpressureDiff_host = 0;
            maxpressureDiff_previous = 0;
            pressureChangeSmallEnough_host = FALSE;
            while (pressureChangeSmallEnough_host == FALSE) {
#endif
	            // do the predictor step (writes to predictor)
    	        cudaVerifyKernel((PredictorStep<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>()));
			    cudaVerify(hipDeviceSynchronize());
            	// get the derivatives at the predictor locations
		        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &predictor_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
		        cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &predictor_pointmass_device, sizeof(struct Pointmass)));
#endif

    	        if (param.selfgravity) {
        	        copy_gravitational_accels_device_to_device(&predictor_device, &p_device);
                }

            	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(substep_currentTimeD), &substep_currentTime, sizeof(double)));
				rightHandSide();
            	// do the corrector step with the predictor and write it to in predictor
	        	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &p_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
	        	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
#if PALPHA_POROSITY
            	cudaVerifyKernel((CorrectorStepPorous<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>()));
				cudaVerify(hipDeviceSynchronize());
				cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &predictor_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
	        	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
				cudaVerifyKernel((calculatePressure<<<numberOfMultiprocessors * 4, NUM_THREADS_PRESSURE>>>()));
    			cudaVerify(hipDeviceSynchronize());
				cudaVerifyKernel((pressureChangeCheck<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>(maxpressureDiffPerBlock)));
				cudaVerify(hipDeviceSynchronize());
				cudaVerify(hipMemcpyFromSymbol(&pressureChangeSmallEnough_host, HIP_SYMBOL(pressureChangeSmallEnough), sizeof(int)));
                cudaVerify(hipMemcpyFromSymbol(&maxpressureDiff_host, HIP_SYMBOL(maxpressureDiff), sizeof(double)));
                if (pressureChangeSmallEnough_host == FALSE) {
                    /* redo predictor step with smaller timestep, derivatives are in p_device */
                    printf("Reducing timestep due to Pressure Check function to: %.17e\n", dt_host);
                    if (fabs(maxpressureDiff_host -maxpressureDiff_previous) < 1e-3) {
                        maxpressureDiff_cnt++;
                    }
                    maxpressureDiff_previous = maxpressureDiff_host;
                    if (maxpressureDiff_cnt > 1) {
                        printf("Cannot reduce timestep anymore, continuing with dt %.17e and maxpressurediff %.17e", dt_host, maxpressureDiff_host);
                        pressureChangeSmallEnough_host = TRUE;
                    }
                }
				if (pressureChangeSmallEnough_host == FALSE) {
                    cudaVerify(hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double)));
					substep_currentTime = currentTime + dt_host * 0.5;
					cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(p), &p_device, sizeof(struct Particle)));
#if GRAVITATING_POINT_MASSES
	        	    cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(pointmass), &pointmass_device, sizeof(struct Pointmass)));
#endif
					printf("Reducing timestep due to Pressure Check function to: %e\n", dt_host);
				} else {
					cudaVerify(hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double)));
					currentTime += dt_host;
					cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(currentTimeD), &currentTime, sizeof(double)));
					copy_particles_variables_device_to_device(&p_device, &predictor_device);
#if GRAVITATING_POINT_MASSES
					copy_pointmass_variables_device_to_device(&pointmass_device, &predictor_pointmass_device);
#endif
					cudaVerify(hipDeviceSynchronize());
				}
			}
#else
            cudaVerifyKernel((CorrectorStep<<<numberOfMultiprocessors, NUM_THREADS_PC_INTEGRATOR>>>()));
			cudaVerify(hipDeviceSynchronize());
#endif
            /* get the time and the time step from the gpu */
            cudaVerify(hipMemcpyFromSymbol(&currentTime, HIP_SYMBOL(currentTimeD), sizeof(double)));
	    //step was successful --> do something (e.g. look for min/max pressure...)
	    afterIntegrationStep();

		} // current time < end time loop
		// write results
#if FRAGMENTATION
        cudaVerify(hipDeviceSynchronize());
        cudaVerifyKernel((damageLimit<<<numberOfMultiprocessors*4, NUM_THREADS_PC_INTEGRATOR>>>()));
        cudaVerify(hipDeviceSynchronize());
#endif
        copyToHostAndWriteToFile(timestep, lastTimestep);

	} // timestep loop

	// free memory

    int free_immutables = 1;
    free_particles_memory(&predictor_device, free_immutables);
#if GRAVITATING_POINT_MASSES
    free_pointmass_memory(&predictor_pointmass_device, free_immutables);
#endif
	cudaVerify(hipFree(courantPerBlock));
	cudaVerify(hipFree(forcesPerBlock));
	cudaVerify(hipFree(dtSPerBlock));
	cudaVerify(hipFree(dtePerBlock));
	cudaVerify(hipFree(dtrhoPerBlock));
	cudaVerify(hipFree(dtdamagePerBlock));
	cudaVerify(hipFree(dtalphaPerBlock));
	cudaVerify(hipFree(dtbetaPerBlock));
	cudaVerify(hipFree(dtartviscPerBlock));
    cudaVerify(hipFree(maxpressureDiffPerBlock));
    cudaVerify(hipFree(dtalpha_epsporPerBlock));
    cudaVerify(hipFree(dtepsilon_vPerBlock));
}
