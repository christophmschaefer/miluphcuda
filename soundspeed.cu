#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "miluph.h"
#include "soundspeed.h"
#include "config_parameter.h"
#include "pressure.h"
#include "aneos.h"


__global__ void calculateSoundSpeed()
{
    register int i, inc, matId;
    int d;
    int j;
    double m_com;
    register double cs, rho, pressure, eta, omega0, z, cs_sq,  cs_c_sq, cs_e_sq, Gamma_e, mu, y; //Gamma_c;
    int i_rho, i_e;

    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (EOS_TYPE_POLYTROPIC_GAS == matEOS[matId]) {
            p.cs[i] = sqrt(matPolytropicK[matId] * pow(p.rho[i], matPolytropicGamma[matId]-1.0));
        } else if (EOS_TYPE_LOCALLY_ISOTHERMAL_GAS == matEOS[matId]) {
            // p = vkep \times scale_height
            double distance = 0.0;
            distance = p.x[i] * p.x[i];
#if DIM > 1
            distance += p.y[i]*p.y[i];
#if DIM > 2
            distance += p.z[i]*p.z[i];
#endif
#endif
            distance = sqrt(distance);
            m_com = 0;
            for (j = 0; j < numPointmasses; j++) {
                m_com += pointmass.m[j];
            }

            double vkep = sqrt(gravConst * m_com/distance);
            p.cs[i] = vkep * scale_height;
        } else if (EOS_TYPE_IDEAL_GAS == matEOS[matId]) {
            p.cs[i] = sqrt(matPolytropicGamma[matId] * p.p[i] / p.rho[i]);
        } else if (EOS_TYPE_TILLOTSON == matEOS[matId]) {
            rho = p.rho[i];
            eta = rho / matTillRho0[matId];
            omega0 = p.e[i]/(matTillE0[matId]*eta*eta) + 1.0;
            pressure = p.p[i];
            mu = eta - 1.0;
            z = (1.0 - eta)/eta;
            //condensed and expanded cold states
            if (eta >= 0.0 || p.e[i] < matTillEiv[matId]) {
                if (pressure < 0.0 || eta < matRhoLimit[matId]) pressure = 0.0;
                cs_sq = matTilla[matId]*p.e[i]+(matTillb[matId]*p.e[i])/(omega0*omega0)*(3.0*omega0-2.0) +
                    (matTillA[matId]+2.0*matTillB[matId]*mu)/rho + pressure/(rho*rho)*(matTilla[matId]*rho+matTillb[matId]*rho/(omega0*omega0));
            }
            //expanded hot states
            else if (p.e[i] > matTillEcv[matId]) {
                Gamma_e = matTilla[matId] + matTillb[matId]/omega0*exp(-matTillBeta[matId]*z*z);
                cs_sq = (Gamma_e+1.0)*pressure/rho+matTillA[matId]/rho*exp(-(matTillAlpha[matId]*z+matTillBeta[matId]*z*z))*(1.0+mu)/(eta*eta)*(matTillAlpha[matId]+2.0*matTillBeta[matId]*z-eta)
                    + matTillb[matId]*rho*p.e[i]/(omega0*omega0*eta*eta)
                    *exp(-matTillBeta[matId]*z*z)*(2.0*matTillBeta[matId]*z*omega0/matTillRho0[matId] + 1.0)/(matTillE0[matId]*rho)*(2.0*p.e[i]-pressure/rho);
            }
            //intermediate states
            else {
                Gamma_e = matTilla[matId] + matTillb[matId]/omega0*exp(-matTillBeta[matId]*z*z);
                cs_e_sq = (Gamma_e+1.0)*pressure/rho+matTillA[matId]/rho*exp(-(matTillAlpha[matId]*z+matTillBeta[matId]*z*z))*(1.0+mu)/(eta*eta)*(matTillAlpha[matId]+2.0*matTillBeta[matId]*z-eta)
                    + matTillb[matId]*rho*p.e[i]/(omega0*omega0*eta*eta)
                    *exp(-matTillBeta[matId]*z*z)*(2.0*matTillBeta[matId]*z*omega0/matTillRho0[matId] + 1.0)/(matTillE0[matId]*rho)*(2.0*p.e[i]-pressure/rho);
                if (pressure < 0.0 || eta < matRhoLimit[matId]) pressure = 0.0;  //set pressure to zero only for condensed state
                cs_c_sq = matTilla[matId]*p.e[i]+(matTillb[matId]*p.e[i])/(omega0*omega0)*(3.0*omega0-2.0) +
                    (matTillA[matId]+2.0*matTillB[matId]*mu)/rho + pressure/(rho*rho)*(matTilla[matId]*rho+matTillb[matId]*rho/(omega0*omega0));
                y = (p.e[i]-matTillEiv[matId])/(matTillEcv[matId]-matTillEiv[matId]);
                cs_sq = cs_e_sq*(1.0-y)+cs_c_sq*y;
            }
            // set to >= lower limit
            if (cs_sq < matcsLimit[matId]*matcsLimit[matId]){
                p.cs[i] = matcsLimit[matId];
            } else {
                p.cs[i] = sqrt(cs_sq);
            }
        } else if (EOS_TYPE_ANEOS == matEOS[matId]) {
            // find array-indices just below the actual values of rho and e
            i_rho = array_index(p.rho[i], aneos_rho_c+aneos_rho_id_c[matId], aneos_n_rho_c[matId]);
            i_e = array_index(p.e[i], aneos_e_c+aneos_e_id_c[matId], aneos_n_e_c[matId]);
            // interpolate (bi)linearly to obtain the sound speed
            p.cs[i] = bilinear_interpolation_from_linearized(p.rho[i], p.e[i], aneos_cs_c+aneos_matrix_id_c[matId], aneos_rho_c+aneos_rho_id_c[matId], aneos_e_c+aneos_e_id_c[matId], i_rho, i_e, aneos_n_rho_c[matId], aneos_n_e_c[matId]);
            // set to >= lower limit
            if (p.cs[i] < matcsLimit[matId]) {
                p.cs[i] = matcsLimit[matId];
            }
#if PALPHA_POROSITY
        } else if (EOS_TYPE_JUTZI_MURNAGHAN == matEOS[matId]) {
            //p.cs[i] = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
//            if (p.alpha_jutzi[i] > 1.0 && abs(p.dalphadp[i]) > 0) {
//                if (abs(p.delpdelrho[i]) > 0.0 || abs(p.delpdele[i]) > 0.0) {
//                    p.cs[i] = sqrt((p.alpha_jutzi[i] * p.delpdelrho[i] + p.delpdele[i] * p.p[i] / (p.rho[i] * p.rho[i]))
//                                / (p.alpha_jutzi[i] + p.dalphadp[i] * (p.p[i] - p.rho[i] * p.delpdelrho[i])));
//                }
//            }
//            if (!isnan(p.cs[i])) {
//                p_rhs.cs_old[i] = p.cs[i];
//            } else {
//                p.cs[i] = p_rhs.cs_old[i];
//            }
            /* switched from jutzis implementation of the soundspeed to a linear soundspeed from cs_porous with alpha=alpha0 to cs_solid with alpha=1 (also done in iSale) */
            p.cs[i] = matcs_solid[matId] + (matcs_porous[matId] - matcs_solid[matId]) * (p.alpha_jutzi[i] - 1.0) / (matporjutzi_alpha_0[matId] - 1.0);
#if DEBUG_MISC
            if (isnan(p.cs[i])) {
                printf("i %d alpha_jutzi %e delpdelrho %e delpdele %e dalphadp %e p %e rho %e\n", i, p.alpha_jutzi[i], p.delpdelrho[i], p.delpdele[i], p.dalphadp[i], p.p[i], p.rho[i]);
                assert(0);
            }
#endif
        } else if (EOS_TYPE_JUTZI_ANEOS == matEOS[matId]) {
            // find array-indices just below the actual values of rho and e
            i_rho = array_index(p.rho[i], aneos_rho_c+aneos_rho_id_c[matId], aneos_n_rho_c[matId]);
            i_e = array_index(p.e[i], aneos_e_c+aneos_e_id_c[matId], aneos_n_e_c[matId]);
            // interpolate (bi)linearly to obtain the sound speed
            cs = bilinear_interpolation_from_linearized(p.rho[i], p.e[i], aneos_cs_c+aneos_matrix_id_c[matId], aneos_rho_c+aneos_rho_id_c[matId], aneos_e_c+aneos_e_id_c[matId], i_rho, i_e, aneos_n_rho_c[matId], aneos_n_e_c[matId]);
            // do interpolation only if computed sound speed is above cs_porous, to capture
            // only compaction process, but not expanded states for example...
            if( cs > matcs_porous[matId] ) {
                // linear interpolation between the sound speed in the matrix (from above) and cs_porous (a constant)
                cs = cs + (matcs_porous[matId] - cs) * (p.alpha_jutzi[i] - 1.0) / (matporjutzi_alpha_0[matId] - 1.0);
            }
            // set to >= lower limit
            if (cs < matcsLimit[matId]) {
                p.cs[i] = matcsLimit[matId];
            } else {
                p.cs[i] = cs;
            }
#if DEBUG_MISC
            if (isnan(p.cs[i])) {
                printf("i %d alpha_jutzi %e delpdelrho %e delpdele %e dalphadp %e p %e rho %e\n", i, p.alpha_jutzi[i], p.delpdelrho[i], p.delpdele[i], p.dalphadp[i], p.p[i], p.rho[i]);
                assert(0);
            }
#endif
        } else if (EOS_TYPE_JUTZI == matEOS[matId]) {
            rho = p.rho[i];
            eta = rho / matTillRho0[matId];
            omega0 = p.e[i]/(matTillE0[matId]*eta*eta) + 1.0;
            pressure = p.p[i];
            mu = eta - 1.0;
            z = (1.0 - eta)/eta;
            //condensed and expanded cold states
            if (eta >= 0.0 || p.e[i] < matTillEiv[matId]) {
                if (pressure < 0.0 || eta < matRhoLimit[matId])
                    pressure = 0.0;
                cs_sq = matTilla[matId]*p.e[i]+(matTillb[matId]*p.e[i])/(omega0*omega0)*(3.0*omega0-2.0) +
                    (matTillA[matId]+2.0*matTillB[matId]*mu)/rho + pressure/(rho*rho)*(matTilla[matId]*rho+matTillb[matId]*rho/(omega0*omega0));
            }
            //expanded hot states
            else if (p.e[i] > matTillEcv[matId]) {
                Gamma_e = matTilla[matId] + matTillb[matId]/omega0*exp(-matTillBeta[matId]*z*z);
                cs_sq = (Gamma_e+1.0)*pressure/rho+matTillA[matId]/rho*exp(-(matTillAlpha[matId]*z+matTillBeta[matId]*z*z))*(1.0+mu)/(eta*eta)*(matTillAlpha[matId]+2.0*matTillBeta[matId]*z-eta)
                    + matTillb[matId]*rho*p.e[i]/(omega0*omega0*eta*eta)
                    *exp(-matTillBeta[matId]*z*z)*(2.0*matTillBeta[matId]*z*omega0/matTillRho0[matId] + 1.0)/(matTillE0[matId]*rho)*(2.0*p.e[i]-pressure/rho);
            }
            //intermediate states
            else {
                Gamma_e = matTilla[matId] + matTillb[matId]/omega0*exp(-matTillBeta[matId]*z*z);
                cs_e_sq = (Gamma_e+1.0)*pressure/rho+matTillA[matId]/rho*exp(-(matTillAlpha[matId]*z+matTillBeta[matId]*z*z))*(1.0+mu)/(eta*eta)*(matTillAlpha[matId]+2.0*matTillBeta[matId]*z-eta)
                    + matTillb[matId]*rho*p.e[i]/(omega0*omega0*eta*eta)
                    *exp(-matTillBeta[matId]*z*z)*(2.0*matTillBeta[matId]*z*omega0/matTillRho0[matId] + 1.0)/(matTillE0[matId]*rho)*(2.0*p.e[i]-pressure/rho);
                if (pressure < 0.0 || eta < matRhoLimit[matId]) pressure = 0.0;  //set pressure to zero only for condensed state
                cs_c_sq = matTilla[matId]*p.e[i]+(matTillb[matId]*p.e[i])/(omega0*omega0)*(3.0*omega0-2.0) +
                    (matTillA[matId]+2.0*matTillB[matId]*mu)/rho + pressure/(rho*rho)*(matTilla[matId]*rho+matTillb[matId]*rho/(omega0*omega0));
                y = (p.e[i]-matTillEiv[matId])/(matTillEcv[matId]-matTillEiv[matId]);
                cs_sq = cs_e_sq*(1.0-y)+cs_c_sq*y;
            }
            // do interpolation only if computed sound speed is above cs_porous, to capture
            // only compaction process, but not expanded states for example...
            if( cs_sq > matcs_porous[matId]*matcs_porous[matId] ) {
                cs = sqrt(cs_sq);
                // linear interpolation between the sound speed in the matrix (from above) and cs_porous (a constant)
                cs = cs + (matcs_porous[matId] - cs) * (p.alpha_jutzi[i] - 1.0) / (matporjutzi_alpha_0[matId] - 1.0);
                // set to >= lower limit
                if (cs < matcsLimit[matId]) {
                    p.cs[i] = matcsLimit[matId];
                } else {
                    p.cs[i] = cs;
                }
            } else {
                // set to >= lower limit
                if (cs_sq < matcsLimit[matId]*matcsLimit[matId]){
                    p.cs[i] = matcsLimit[matId];
                } else {
                    p.cs[i] = sqrt(cs_sq);
                }
            }
#if DEBUG_MISC
            if (isnan(p.cs[i])) {
                printf("i %d alpha_jutzi %e delpdelrho %e delpdele %e dalphadp %e p %e rho %e\n", i, p.alpha_jutzi[i], p.delpdelrho[i], p.delpdele[i], p.dalphadp[i], p.p[i], p.rho[i]);
                assert(0);
            }
#endif
#endif // PALPHA_POROSITY
#if SIRONO_POROSITY
        } else if (EOS_TYPE_SIRONO == matEOS[matId]) {
            if (p.flag_plastic[i] > 0)
                p.cs[i] = sqrt(p.compressive_strength[i] / p.rho[i]);
            else
                p.cs[i] = sqrt(p.K[i] / p.rho_0prime[i]);
#endif
#if EPSALPHA_POROSITY
        /* Improvements to epsilon-alpha model by Collins et al 2010 */
        } else if (EOS_TYPE_EPSILON == matEOS[matId]) {
            double c_s0 = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
            double c_p0 = sqrt(matBulkmodulus[matId]/(matTillRho0[matId] / matporepsilon_alpha_0[matId]));
            p.cs[i] = c_s0 + (p.alpha_epspor[i] - 1.0) / (matporepsilon_alpha_0[matId] - 1.0) * (c_p0 - c_s0);
#endif
        }
        // other material types have a constant soundspeed which is set in initializeSoundspeed()
    }
}



__global__ void initializeSoundspeed()
{
    register int i, inc, matId;
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (EOS_TYPE_POLYTROPIC_GAS == matEOS[matId]) {
            p.cs[i] = 0.0; // for gas this will be calculated each step by kernel calculateSoundSpeed
        } else if (EOS_TYPE_ISOTHERMAL_GAS == matEOS[matId]) {
            /* this is pure molecular hydrogen at 10 K */
            p.cs[i] = 203.0;
        } else if (EOS_TYPE_TILLOTSON == matEOS[matId]) {
            p.cs[i] = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
        } else if (EOS_TYPE_ANEOS == matEOS[matId]) {
            p.cs[i] = aneos_bulk_cs_c[matId];
        } else if (EOS_TYPE_MURNAGHAN == matEOS[matId]) {
            p.cs[i] = sqrt(matBulkmodulus[matId]/matRho0[matId]);
        } else if (EOS_TYPE_JUTZI == matEOS[matId]) {
            p.cs[i] = matcs_porous[matId];
        } else if (EOS_TYPE_JUTZI_ANEOS == matEOS[matId]) {
            p.cs[i] = matcs_porous[matId];
        } else if (EOS_TYPE_JUTZI_MURNAGHAN == matEOS[matId]) {
            p.cs[i] = matcs_porous[matId];
        } else if (EOS_TYPE_REGOLITH == matEOS[matId]) {
            //sound speed in soil is typically between 450 and 600 m/s according to Ha H. Bui 2008
            p.cs[i] = 500.0;
//        } else if (EOS_TYPE_EPSILON == matEOS[matId]) {
//            p.cs[i] = sqrt(matBulkmodulus[matId]/matTillRho0[matId]);
        }
    }
}
