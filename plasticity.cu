#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */



#include "plasticity.h"
#include "config_parameter.h"
#include "parameter.h"
#include "miluph.h"
#include "pressure.h"
#include "float.h"

#if SOLID
__global__ void plasticity()
{
    register int i, inc, matId, d, e;
    register double alpha_phi, kc, I1, sqrt_J2, rn;
#if DIM == 2
    register double shear, bulk, poissons_ratio, sz;
#endif
    register double S_i[DIM][DIM];
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        matId = p_rhs.materialId[i];
        if (EOS_TYPE_REGOLITH == matEOS[matId]) {

            alpha_phi = matAlphaPhi[matId];
            kc = matCohesionCoefficient[matId];


#if DIM == 2
            shear = matShearmodulus[matId];
            bulk = matBulkmodulus[matId];
            poissons_ratio = (3*bulk - 2*shear) / (2*(3*bulk + shear));
            I1 = (1 + poissons_ratio) * (p.S[stressIndex(i, 0, 0)] + p.S[stressIndex(i, 1, 1)]);
#else
            I1 = p.S[stressIndex(i,0,0)] + p.S[stressIndex(i,1,1)] + p.S[stressIndex(i,2,2)];
#endif


            //Tension cracking treatment
            //Equation 29, Bui et al., 2008
            if (-I1*alpha_phi + kc < 0) {
                for (d = 0; d < DIM; d++) {
                    p.S[stressIndex(i, d, d)] -= (I1 - kc/alpha_phi)/3.0;
                }
            }

#if DIM == 2
            I1 = (1 + poissons_ratio) * (p.S[stressIndex(i, 0, 0)] + p.S[stressIndex(i, 1, 1)]);
#else
            I1 = p.S[stressIndex(i,0,0)] + p.S[stressIndex(i,1,1)] + p.S[stressIndex(i,2,2)];
#endif


            //get S
            for (d = 0; d < DIM; d++) {
                for (e = 0; e < DIM; e++) {
                    S_i[d][e] = p.S[stressIndex(i, d, e)];
                }
                S_i[d][d] -= I1/3.0;
            }
#if DIM == 2
            sz = poissons_ratio*(S_i[0][0] + S_i[1][1]);
#endif

            sqrt_J2 = 0.0;
            for (d = 0; d < DIM; d++) {
                for (e = 0; e < DIM; e++) {
                    sqrt_J2 += S_i[d][e]*S_i[d][e];
                }
            }
#if DIM == 2
            sqrt_J2 += sz*sz;
#endif
            sqrt_J2 *= 0.5;
            sqrt_J2 = sqrt(sqrt_J2);


            //stress-scaling
            //Equation 31, Bui et al., 2008
            if (sqrt_J2 > 0) {
                rn = (-I1*alpha_phi + kc) / sqrt_J2;
                rn = min(rn, 1.0);
                for (d = 0; d < DIM; d++) {
                    for (e = 0; e < DIM; e++) {
                        p.S[stressIndex(i, d, e)] = rn*S_i[d][e];
                    }
                    p.S[stressIndex(i, d, d)] += I1/3.0;
                }
            }


        } //end if (EOS_TYPE_REGOLITH)
    }
}
#endif

#if SOLID
__global__ void vonMisesPlasticity(void) {
    // introduce plastic behaviour by limiting the deviatoric stress
    register int i, inc, d, e;
    register double mises_f, tmp;
    register double I1, J2, sqrt_J2;
    register double y, y_i, y_M, y_0, y_0_d, ytmp;
    register double damage;
    /* drucker prager constants */
    register double A, B;
    double mu_i, mu_d; // coefficients of internal friction

    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
        // VISCOUS_REGOLITH is treated in timeintegration.cu when \sigma is calculated
        if (matEOS[p_rhs.materialId[i]] == EOS_TYPE_VISCOUS_REGOLITH) {
            continue;
        }

        /* second invariant of the stress tensor */
        J2 = 0;
        mises_f = 0;

        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
                tmp = p.S[stressIndex(i, d, e)];
                J2 += tmp*tmp;
            }
        }

        J2 *= 0.5;
        sqrt_J2 = sqrt(J2);

        /* first invariant of the stress tensor */
        I1 = -3.0 * p.p[i];


#if MOHR_COULOMB_PLASTICITY
        // mohr coulomb yield criterion
        // matInternalFriction = \mu = tan(matFrictionAngle)
        y = matInternalFriction[p_rhs.materialId[i]] * p.p[i] + matCohesion[p_rhs.materialId[i]];
        // drucker prager like -> compare to sqrt(J2)
        if (J2 > 0) {
            mises_f = y/sqrt_J2;
        }

        if (mises_f > 1)
            mises_f = 1;
#elif DRUCKER_PRAGER_PLASTICITY
        A = B = 0;
        // drucker prager constants from mohr-coulomb constants -> 3D!
        A = 6. * matCohesion[p_rhs.materialId[i]] * cos(matFrictionAngle[p_rhs.materialId[i]])
                / (sqrt(3.) * (3. - sin(matFrictionAngle[p_rhs.materialId[i]])));
        B = 2. * sin(matFrictionAngle[p_rhs.materialId[i]]) / (sqrt(3.) * (3. - sin(matFrictionAngle[p_rhs.materialId[i]])));

        // yield strength determined by drucker prager condition
        y = A + 3.0*p.p[i] * B;
        // drucker prager like -> compare to sqrt(J2)
        if (J2 > 0) {
            mises_f = y/sqrt_J2;
        }

        if (mises_f > 1)
            mises_f = 1;
#elif COLLINS_PRESSURE_DEPENDENT_YIELD_STRENGTH
        y_0 = matCohesion[p_rhs.materialId[i]];
        y_0_d = matCohesionDamaged[p_rhs.materialId[i]];
        y_M = matYieldStress[p_rhs.materialId[i]];
        mu_i = matInternalFriction[p_rhs.materialId[i]];
# if FRAGMENTATION
        mu_d = matInternalFrictionDamaged[p_rhs.materialId[i]];
# endif
        // shear strength of the intact material
        ytmp = y_0;
        if (p.p[i] > 0) {
            ytmp += mu_i * p.p[i]
                / (1 + mu_i * p.p[i]  / (y_M - y_0) );
        } else {
            ytmp = y_0;
        }
# if FRAGMENTATION
        damage = p.damage_total[i];
        if (damage > 1) damage = 1.0;
        // yield strength of damaged material
        if (p.p[i] > 0) {
            y = y_0_d + mu_d * p.p[i];
            /* limit the yield strength of damaged material to the yield strength of intact material */
            if (damage < 1) {
                y = (1-damage) * ytmp + damage*y;
                if (y > ytmp) y = ytmp;
            }
        } else {
            y = y_0;
        }
# else
        y = ytmp;
# endif
        // drucker prager like -> compare to sqrt(J2)
        if (J2 > 0) {
            mises_f = y/sqrt_J2;
        }

        if (mises_f > 1)
            mises_f = 1;
#else // simple von Mises yield criterion without *any* dependency
        y = matYieldStress[p_rhs.materialId[i]];
#if SIRONO_POROSITY
        // Shear Strength using Sironos Model
        if (matEOS[p_rhs.materialId[i]] == EOS_TYPE_SIRONO) {
            y = sqrt((-1.0) * p.tensile_strength[i] * p.compressive_strength[i]);
            p.shear_strength[i] = y;
        } else {
            p.shear_strength[i] = DBL_MAX;
            y = p.shear_strength[i];
        }
#endif
        // von mises limit like
        if (J2 > 0) {
            mises_f = y*y/(3*J2);
        }

        if (mises_f > 1)
            mises_f = 1;
#endif

        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
                p.S[stressIndex(i, d, e)] *= mises_f;
            }
        }
    }
}

#endif





#if SOLID
#if JC_PLASTICITY
__global__ void JohnsonCookPlasticity(void) {
    // introduce plastic behaviour by limiting the deviatoric stress
    register int i, inc, d, e;
    register double J2, jc_f, y_0, tmp;

    register double y_jc = 0;
    register double T_star = 0;
    register double B, n, m, edot0, C, Tref, Tmelt;
    /*register double Cp, CV;*/

    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {

        J2 = 0;
        jc_f = 0;
        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
                tmp = p.S[stressIndex(i, d, e)];
                J2 += tmp*tmp;
            }
        }


        y_0 = matjc_y0[p_rhs.materialId[i]];
        B = matjc_B[p_rhs.materialId[i]];
        n = matjc_n[p_rhs.materialId[i]];
        m = matjc_m[p_rhs.materialId[i]];
        edot0 = matjc_edot0[p_rhs.materialId[i]];
        C = matjc_C[p_rhs.materialId[i]];
        Tref = matjc_Tref[p_rhs.materialId[i]];
        Tmelt = matjc_Tmelt[p_rhs.materialId[i]];
        /*Cp = matCp[p_rhs.materialId[i]];*/
        /*CV = matCV[p_rhs.materialId[i]];*/

        register double edotp = p.edotp[i];
        register double ep = p.ep[i];
        register double T = p.T[i];

        // T_star has to be different for different cases, otherwise we have complex numbers and nans
        if (T < Tref) {
            T_star = 0;
        } else if (T > Tmelt) {
            T_star = 1;
        } else {
            T_star = (T - Tref) / (Tmelt - Tref);
        }


        // Calculating flow stress according to Johnson and Cook
        if (edotp > 0) {
            y_jc = (y_0 + B*(pow(ep,n))) * (1 + C*log(edotp / edot0)) * (1 - pow(T_star,m));
        } else {
            y_jc = y_0;
        }


        y_jc = y_jc * y_jc;
        J2 = J2 * 1.5;
        if (J2 > y_jc)
            jc_f = y_jc/J2;
        else
            jc_f = 1;
        for (d = 0; d < DIM; d++) {
            for (e = 0; e < DIM; e++) {
                p.S[stressIndex(i, d, e)] *= jc_f;
            }
        }

        /* remember for calculation of edotp later on */
        p.jc_f[i] = jc_f;
        p.edotp[i] = 0.0;
    }

}

#endif // jc_plasticity
#endif
