#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "miluph.h"
#include "memory_handling.h"
#include "aneos.h"


/* allocate memory on the device for pointmasses */
int allocate_pointmass_memory(struct Pointmass *a, int allocate_immutables)
{
    int rc = 0;

	cudaVerify(hipMalloc((void**)&a->x, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->vx, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->ax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feedback_ax, memorySizeForPointmasses));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->y, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->vy, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->ay, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feedback_ay, memorySizeForPointmasses));
# if DIM > 2
	cudaVerify(hipMalloc((void**)&a->z, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->vz, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->az, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feedback_az, memorySizeForPointmasses));
# endif
#endif
	cudaVerify(hipMalloc((void**)&a->m, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->rmin, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->rmax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feels_particles, integermemorySizeForPointmasses));

    return rc;
}



/* allocate memory on the device for particles */
int allocate_particles_memory(struct Particle *a, int allocate_immutables)
{
    int rc = 0;

#if TENSORIAL_CORRECTION
    // also moved to p_device only
//	cudaVerify(hipMalloc((void**)&a->tensorialCorrectionMatrix, memorySizeForStress));
    // not needed anymore, let's save memory --- tschakka!
/*    if (allocate_immutables) {
        cudaVerify(hipMalloc((void**)&a->tensorialCorrectiondWdrr, MAX_NUM_INTERACTIONS * maxNumberOfParticles * sizeof(double)));
    } */
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipMalloc((void**)&a->dedt, memorySizeForParticles));
#endif

#if DISPH
    cudaVerify(hipMalloc((void**)&a->dUdt, memorySizeForParticles));
#endif

#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipMalloc((void**)&a->muijmax, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&a->drhodt, memorySizeForParticles));

#if SOLID
	cudaVerify(hipMalloc((void**)&a->S, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&a->dSdt, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&a->local_strain, memorySizeForParticles));
#endif

#if NAVIER_STOKES
	cudaVerify(hipMalloc((void**)&a->Tshear, memorySizeForStress));
#endif

#if INVISCID_SPH
	cudaVerify(hipMalloc((void**)&a->beta, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->beta_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->divv_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dbetadt, memorySizeForParticles));
#endif

#if FRAGMENTATION
	memorySizeForActivationThreshold = maxNumberOfParticles * MAX_NUM_FLAWS * sizeof(double);
	cudaVerify(hipMalloc((void**)&a->d, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->damage_total, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dddt, memorySizeForParticles));

	cudaVerify(hipMalloc((void**)&a->numFlaws, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&a->numActiveFlaws, memorySizeForInteractions));
    if (allocate_immutables) {
	    cudaVerify(hipMalloc((void**)&a->flaws, memorySizeForActivationThreshold));
    }
#if PALPHA_POROSITY
	cudaVerify(hipMalloc((void**)&a->damage_porjutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->ddamage_porjutzidt, memorySizeForParticles));
#endif
#endif

    if (allocate_immutables) {
        cudaVerify(hipMalloc((void**)&a->h0, memorySizeForParticles));
    }

#if GHOST_BOUNDARIES
	cudaVerify(hipMalloc((void**)&a->real_partner, memorySizeForInteractions));
#endif

#if PALPHA_POROSITY
	cudaVerify(hipMalloc((void**)&a->pold, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->alpha_jutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->alpha_jutzi_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dalphadt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dalphadp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dalphadrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->delpdelrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->delpdele, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->f, memorySizeForParticles));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipMalloc((void**)&a->compressive_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->tensile_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->shear_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->K, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_0prime, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_c_plus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_c_minus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->flag_rho_0prime, memorySizeForInteractions));
    cudaVerify(hipMalloc((void**)&a->flag_plastic, memorySizeForInteractions));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipMalloc((void**)&a->alpha_epspor, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->dalpha_epspordt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->epsilon_v, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->depsilon_vdt, memorySizeForParticles));
#endif

    cudaVerify(hipMalloc((void**)&a->x0, memorySizeForTree));
#if DIM > 1
    cudaVerify(hipMalloc((void**)&a->y0, memorySizeForTree));
#if DIM > 2
    cudaVerify(hipMalloc((void**)&a->z0, memorySizeForTree));
#endif
#endif
	cudaVerify(hipMalloc((void**)&a->x, memorySizeForTree));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->y, memorySizeForTree));
#endif
	cudaVerify(hipMalloc((void**)&a->vx, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->vy, memorySizeForParticles));
#endif
	cudaVerify(hipMalloc((void**)&a->dxdt, memorySizeForParticles));
#if DIM > 1
 	cudaVerify(hipMalloc((void**)&a->dydt, memorySizeForParticles));
#endif

#if XSPH
	cudaVerify(hipMalloc((void**)&a->xsphvx, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->xsphvy, memorySizeForParticles));
#endif
#endif
	cudaVerify(hipMalloc((void**)&a->ax, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->g_ax, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->ay, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->g_ay, memorySizeForParticles));
#endif
	cudaVerify(hipMalloc((void**)&a->m, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&a->h, memorySizeForParticles));
#if INTEGRATE_SML
	cudaVerify(hipMalloc((void**)&a->dhdt, memorySizeForParticles));
#endif

#if SML_CORRECTION
	cudaVerify(hipMalloc((void**)&a->sml_omega, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&a->rho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->p, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->e, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->cs, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->noi, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&a->depth, memorySizeForInteractions));

#if DISPH
    cudaVerify(hipMalloc((void**)&a->q, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->U, memorySizeForParticles));
#endif

#if MORE_OUTPUT
	cudaVerify(hipMalloc((void**)&a->p_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->p_max, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->e_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->e_max, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->cs_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->cs_max, memorySizeForParticles));
#endif
// moved to p_device only, so we don't need mem here anymore
//	cudaVerify(hipMalloc((void**)&a->materialId, memorySizeForInteractions));

#if JC_PLASTICITY
	cudaVerify(hipMalloc((void**)&a->ep, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->edotp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->T, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dTdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->jc_f, memorySizeForParticles));
#endif

#if DIM > 2
	cudaVerify(hipMalloc((void**)&a->z, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&a->dzdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->vz, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->az, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->g_az, memorySizeForParticles));
#if XSPH
	cudaVerify(hipMalloc((void**)&a->xsphvz, memorySizeForParticles));
#endif
#endif
	cudaVerify(hipMemset(a->ax, 0, memorySizeForParticles));
	cudaVerify(hipMemset(a->g_ax, 0, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMemset(a->ay, 0, memorySizeForParticles));
	cudaVerify(hipMemset(a->g_ay, 0, memorySizeForParticles));
#if DIM == 3
	cudaVerify(hipMemset(a->az, 0, memorySizeForParticles));
	cudaVerify(hipMemset(a->g_az, 0, memorySizeForParticles));
#endif
#endif

    return rc;
}



int copy_gravitational_accels_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;
    cudaVerify(hipMemcpy(dst->g_ax, src->g_ax, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->g_ay, src->g_ay, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if DIM > 2
    cudaVerify(hipMemcpy(dst->g_az, src->g_az, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

    return rc;
}



int copy_pointmass_derivatives_device_to_device(struct Pointmass *dst, struct Pointmass *src)
{
    int rc = 0;
    cudaVerify(hipMemcpy(dst->ax, src->ax, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vx, src->vx, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->feedback_ax, src->feedback_ax, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->ay, src->ay, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vy, src->vy, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->feedback_ay, src->feedback_ay, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# if DIM > 2
    cudaVerify(hipMemcpy(dst->az, src->az, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vz, src->vz, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->feedback_az, src->feedback_az, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# endif
#endif

    return rc;
}



int copy_particles_derivatives_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy(dst->ax, src->ax, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->g_ax, src->g_ax, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dxdt, src->dxdt, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if DIM > 1
    cudaVerify(hipMemcpy(dst->ay, src->ay, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->g_ay, src->g_ay, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dydt, src->dydt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if DIM > 2
    cudaVerify(hipMemcpy(dst->az, src->az, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->g_az, src->g_az, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dzdt, src->dzdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

    cudaVerify(hipMemcpy(dst->drhodt, src->drhodt, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if INTEGRATE_SML
    cudaVerify(hipMemcpy(dst->dhdt, src->dhdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if SML_CORRECTION
    cudaVerify(hipMemcpy(dst->sml_omega, src->sml_omega, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if PALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->dalphadt, src->dalphadt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->ddamage_porjutzidt, src->ddamage_porjutzidt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->dalpha_epspordt, src->dalpha_epspordt, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->depsilon_vdt, src->depsilon_vdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if INTEGRATE_ENERGY
    cudaVerify(hipMemcpy(dst->dedt, src->dedt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if DISPH
    cudaVerify(hipMemcpy(dst->dUdt, src->dUdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if SOLID
    cudaVerify(hipMemcpy(dst->dSdt, src->dSdt, memorySizeForStress, hipMemcpyDeviceToDevice));
#endif

#if INVISCID_SPH
	cudaVerify(hipMemcpy(dst->dbetadt, src->dbetadt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if JC_PLASTICITY
    cudaVerify(hipMemcpy(dst->edotp, src->edotp, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dTdt, src->dTdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->dddt, src->dddt, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->numActiveFlaws, src->numActiveFlaws, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#endif

    return rc;
}



int copy_pointmass_immutables_device_to_device(struct Pointmass *dst, struct Pointmass *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy((*dst).m, (*src).m, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).feels_particles, (*src).feels_particles, integermemorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).rmin, (*src).rmin, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).rmax, (*src).rmax, memorySizeForPointmasses, hipMemcpyDeviceToDevice));

    return rc;
}



int copy_particles_immutables_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy((*dst).x0, (*src).x0, memorySizeForTree, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy((*dst).y0, (*src).y0, memorySizeForTree, hipMemcpyDeviceToDevice));
#endif
#if DIM > 2
    cudaVerify(hipMemcpy((*dst).z0, (*src).z0, memorySizeForTree, hipMemcpyDeviceToDevice));
#endif
    cudaVerify(hipMemcpy((*dst).m, (*src).m, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).h, (*src).h, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).cs, (*src).cs, memorySizeForParticles, hipMemcpyDeviceToDevice));
    //cudaVerify(hipMemcpy((*dst).materialId, (*src).materialId, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#if FRAGMENTATION
	cudaVerify(hipMemcpy(dst->numFlaws, src->numFlaws, memorySizeForInteractions, hipMemcpyDeviceToDevice));
    //cudaVerify(hipMemcpy(dst->flaws, src->flaws, memorySizeForActivationThreshold, hipMemcpyDeviceToDevice));
#endif

    return rc;
}



int copy_pointmass_variables_device_to_device(struct Pointmass *dst, struct Pointmass *src)
{
    int rc = 0;
    cudaVerify(hipMemcpy(dst->x, src->x, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    // mass is variable
    cudaVerify(hipMemcpy(dst->m, src->m, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vx, src->vx, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->y, src->y, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vy, src->vy, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# if DIM > 2
    cudaVerify(hipMemcpy(dst->z, src->z, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vz, src->vz, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# endif
#endif

    return rc;
}



int copy_particles_variables_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy(dst->x, src->x, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->x0, src->x0, memorySizeForTree, hipMemcpyDeviceToDevice));
    // materialId moved to p_device aka p_rhs only
    //cudaVerify(hipMemcpy((*dst).materialId, (*src).materialId, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->y, src->y, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->y0, src->y0, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vy, src->vy, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#if DIM > 2
    cudaVerify(hipMemcpy(dst->z0, src->z0, memorySizeForTree, hipMemcpyDeviceToDevice));
#endif

    cudaVerify(hipMemcpy(dst->vx, src->vx, memorySizeForParticles, hipMemcpyDeviceToDevice));

    cudaVerify(hipMemcpy(dst->rho, src->rho, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if DISPH
    cudaVerify(hipMemcpy(dst->q, src->q, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->U, src->U, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

    cudaVerify(hipMemcpy(dst->h, src->h, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if INTEGRATE_ENERGY
    cudaVerify(hipMemcpy(dst->e, src->e, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if PALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->alpha_jutzi, src->alpha_jutzi, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->alpha_jutzi_old, src->alpha_jutzi, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dalphadp, src->dalphadp, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dalphadrho, src->dalphadrho, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dp, src->dp, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->delpdelrho, src->delpdelrho, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->delpdele, src->delpdele, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->f, src->f, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->p, src->p, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->pold, src->pold, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->damage_porjutzi, src->damage_porjutzi, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

#if MORE_OUTPUT
    cudaVerify(hipMemcpy(dst->p_min, src->p_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->p_max, src->p_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_min, src->rho_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_max, src->rho_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->e_min, src->e_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->e_max, src->e_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->cs_min, src->cs_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->cs_max, src->cs_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipMemcpy(dst->compressive_strength, src->compressive_strength, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->tensile_strength, src->tensile_strength, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->shear_strength, src->shear_strength, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->K, src->K, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_0prime, src->rho_0prime, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_c_plus, src->rho_c_plus, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_c_minus, src->rho_c_minus, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->flag_rho_0prime, src->flag_rho_0prime, memorySizeForInteractions, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->flag_plastic, src->flag_plastic, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->alpha_epspor, src->alpha_epspor, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dalpha_epspordt, src->dalpha_epspordt, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->epsilon_v, src->epsilon_v, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->depsilon_vdt, src->depsilon_vdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if DIM > 2
    cudaVerify(hipMemcpy(dst->z, src->z, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vz, src->vz, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#if SOLID
    cudaVerify(hipMemcpy(dst->S, src->S, memorySizeForStress, hipMemcpyDeviceToDevice));
#endif
#if NAVIER_STOKES
    cudaVerify(hipMemcpy(dst->Tshear, src->Tshear, memorySizeForStress, hipMemcpyDeviceToDevice));
#endif

#if INVISCID_SPH
    cudaVerify(hipMemcpy(dst->beta, src->beta, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->beta_old, src->beta_old, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->divv_old, src->divv_old, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if JC_PLASTICITY
    cudaVerify(hipMemcpy(dst->ep, src->ep, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->T, src->T, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->jc_f, src->jc_f, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->d, src->d, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->damage_total, src->damage_total, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->numActiveFlaws, src->numActiveFlaws, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#endif

    return rc;
}



/* free runge-kutta memory for pointmasses on the device */
int free_pointmass_memory(struct Pointmass *a, int free_immutables)
{
    int rc = 0;
	cudaVerify(hipFree(a->x));
	cudaVerify(hipFree(a->vx));
	cudaVerify(hipFree(a->ax));
	cudaVerify(hipFree(a->feedback_ax));
	cudaVerify(hipFree(a->m));
	cudaVerify(hipFree(a->feels_particles));
	cudaVerify(hipFree(a->rmin));
	cudaVerify(hipFree(a->rmax));
#if DIM > 1
	cudaVerify(hipFree(a->y));
	cudaVerify(hipFree(a->vy));
	cudaVerify(hipFree(a->ay));
	cudaVerify(hipFree(a->feedback_ay));
# if DIM > 2
	cudaVerify(hipFree(a->z));
	cudaVerify(hipFree(a->vz));
	cudaVerify(hipFree(a->az));
	cudaVerify(hipFree(a->feedback_az));
# endif
#endif

    return rc;
}



/* free runge-kutta memory on the device */
int free_particles_memory(struct Particle *a, int free_immutables)
{
    int rc = 0;

	cudaVerify(hipFree(a->x));
	cudaVerify(hipFree(a->x0));
	cudaVerify(hipFree(a->dxdt));
	cudaVerify(hipFree(a->vx));
	cudaVerify(hipFree(a->ax));
	cudaVerify(hipFree(a->g_ax));
	cudaVerify(hipFree(a->m));
#if DIM > 1
	cudaVerify(hipFree(a->dydt));
	cudaVerify(hipFree(a->y));
	cudaVerify(hipFree(a->y0));
	cudaVerify(hipFree(a->vy0));
	cudaVerify(hipFree(a->vy));
	cudaVerify(hipFree(a->ay));
	cudaVerify(hipFree(a->g_ay));
#endif

#if XSPH
	cudaVerify(hipFree(a->xsphvx));
#if DIM > 1
	cudaVerify(hipFree(a->xsphvy));
#endif
#endif
	cudaVerify(hipFree(a->h));
	cudaVerify(hipFree(a->rho));
	cudaVerify(hipFree(a->p));
	cudaVerify(hipFree(a->e));
	cudaVerify(hipFree(a->cs));
	cudaVerify(hipFree(a->noi));
	cudaVerify(hipFree(a->depth));

#if DISPH
    cudaVerify(hipFree(a->q));
    cudaVerify(hipFree(a->U));
#endif

#if MORE_OUTPUT
	cudaVerify(hipFree(a->p_min));
	cudaVerify(hipFree(a->p_max));
	cudaVerify(hipFree(a->rho_min));
	cudaVerify(hipFree(a->rho_max));
	cudaVerify(hipFree(a->e_min));
	cudaVerify(hipFree(a->e_max));
	cudaVerify(hipFree(a->cs_min));
	cudaVerify(hipFree(a->cs_max));
#endif
    // materialId only on p_device
	//cudaVerify(hipFree(a->materialId));
#if DIM > 2
	cudaVerify(hipFree(a->z));
	cudaVerify(hipFree(a->z0));
	cudaVerify(hipFree(a->dzdt));
	cudaVerify(hipFree(a->vz));
#if XSPH
	cudaVerify(hipFree(a->xsphvz));
#endif
	cudaVerify(hipFree(a->az));
	cudaVerify(hipFree(a->g_az));
#endif


#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipFree(a->muijmax));
#endif
#if (NAVIER_STOKES || BALSARA_SWITCH || INVISCID_SPH || INTEGRATE_ENERGY)
	cudaVerify(hipFree(a->divv));
	cudaVerify(hipFree(a->curlv));
#endif

#if INVISCID_SPH
	cudaVerify(hipFree(a->beta));
	cudaVerify(hipFree(a->beta_old));
	cudaVerify(hipFree(a->divv_old));
	cudaVerify(hipFree(a->dbetadt));
#endif

#if TENSORIAL_CORRECTION
	//cudaVerify(hipFree(a->tensorialCorrectionMatrix));
    /*
    if (free_immutables) {
	    cudaVerify(hipFree(a->tensorialCorrectiondWdrr));
    } */
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipFree(a->dedt));
#endif

#if DISPH
    cudaVerify(hipFree(a->dUdt));
#endif

#if GHOST_BOUNDARIES
	cudaVerify(hipFree(a->real_partner));
#endif

	cudaVerify(hipFree(a->drhodt));

#if INTEGRATE_SML
	cudaVerify(hipFree(a->dhdt));
#endif

#if SML_CORRECTION
    cudaVerify(hipFree(a->sml_omega));
#endif

#if SOLID
	cudaVerify(hipFree(a->S));
	cudaVerify(hipFree(a->dSdt));
	cudaVerify(hipFree(a->local_strain));
#endif
#if NAVIER_STOKES
	cudaVerify(hipFree(a->Tshear));
#endif

#if JC_PLASTICITY
	cudaVerify(hipFree(a->ep));
	cudaVerify(hipFree(a->edotp));
	cudaVerify(hipFree(a->T));
	cudaVerify(hipFree(a->dTdt));
	cudaVerify(hipFree(a->jc_f));
#endif

#if PALPHA_POROSITY
	cudaVerify(hipFree(a->pold));
	cudaVerify(hipFree(a->alpha_jutzi));
	cudaVerify(hipFree(a->alpha_jutzi_old));
	cudaVerify(hipFree(a->dalphadt));
	cudaVerify(hipFree(a->f));
	cudaVerify(hipFree(a->dalphadp));
	cudaVerify(hipFree(a->dp));
	cudaVerify(hipFree(a->delpdelrho));
	cudaVerify(hipFree(a->delpdele));
	cudaVerify(hipFree(a->dalphadrho));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipFree(a->compressive_strength));
    cudaVerify(hipFree(a->tensile_strength));
    cudaVerify(hipFree(a->shear_strength));
    cudaVerify(hipFree(a->K));
    cudaVerify(hipFree(a->rho_0prime));
    cudaVerify(hipFree(a->rho_c_plus));
    cudaVerify(hipFree(a->rho_c_minus));
    cudaVerify(hipFree(a->flag_rho_0prime));
    cudaVerify(hipFree(a->flag_plastic));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipFree(a->alpha_epspor));
    cudaVerify(hipFree(a->dalpha_epspordt));
    cudaVerify(hipFree(a->epsilon_v));
    cudaVerify(hipFree(a->depsilon_vdt));
#endif

#if FRAGMENTATION
	cudaVerify(hipFree(a->d));
	cudaVerify(hipFree(a->damage_total));
	cudaVerify(hipFree(a->dddt));
	cudaVerify(hipFree(a->numFlaws));
	cudaVerify(hipFree(a->numActiveFlaws));
    if (free_immutables) {
	    cudaVerify(hipFree(a->flaws));
    }
    if (free_immutables) {
	    cudaVerify(hipFree(a->h0));
    }
#if PALPHA_POROSITY
	cudaVerify(hipFree(a->damage_porjutzi));
	cudaVerify(hipFree(a->ddamage_porjutzidt));
#endif
#endif

    return rc;
}



/* allocate memory for tree and basic particle struct */
int init_allocate_memory(void)
{
    int rc = 0;

	numberOfNodes = ceil(2.5 * maxNumberOfParticles);
    if (numberOfNodes < 1024*numberOfMultiprocessors)
        numberOfNodes = 1024*numberOfMultiprocessors;

#define WARPSIZE 32
    
    while ((numberOfNodes & (WARPSIZE-1)) != 0)
        numberOfNodes++;

	if (param.verbose) {
        printf("allocating memory for %d particles ...\n", numberOfParticles);
	    printf("allocating memory for %d pointmasses...\n", numberOfPointmasses);
        fprintf(stdout, "Number of nodes of tree: %d\n", numberOfNodes);
    }

	memorySizeForParticles = maxNumberOfParticles * sizeof(double);
	memorySizeForPointmasses = numberOfPointmasses * sizeof(double);
	integermemorySizeForPointmasses = numberOfPointmasses * sizeof(int);
	memorySizeForTree = numberOfNodes * sizeof(double);
	memorySizeForStress = maxNumberOfParticles * DIM * DIM * sizeof(double);
	memorySizeForChildren = numberOfChildren * (numberOfNodes-numberOfRealParticles) * sizeof(int);
	memorySizeForInteractions = maxNumberOfParticles * sizeof(int);

    cudaVerify(hipHostMalloc((void**)&p_host.x, memorySizeForTree));
	cudaVerify(hipHostMalloc((void**)&p_host.vx, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.ax, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.g_ax, memorySizeForParticles));
#if DIM > 1
    cudaVerify(hipHostMalloc((void**)&p_host.y, memorySizeForTree));
	cudaVerify(hipHostMalloc((void**)&p_host.vy, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.ay, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.g_ay, memorySizeForParticles));
#endif
#if DIM > 2
    cudaVerify(hipHostMalloc((void**)&p_host.z, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vz, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.az, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.g_az, memorySizeForParticles));
#endif
    cudaVerify(hipHostMalloc((void**)&p_host.m, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.h, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.p, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.e, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.cs, memorySizeForParticles));

#if DISPH
    cudaVerify(hipHostMalloc((void**)&p_host.q, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.U, memorySizeForParticles));
#endif

#if GRAVITATING_POINT_MASSES
	cudaVerify(hipHostMalloc((void**)&pointmass_host.x, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.vx, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.ax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.x, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.vx, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.ax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feedback_ax, memorySizeForPointmasses));
#if DIM > 1
	cudaVerify(hipHostMalloc((void**)&pointmass_host.y, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.vy, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.ay, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.y, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.vy, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.ay, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feedback_ay, memorySizeForPointmasses));
#if DIM > 2
	cudaVerify(hipHostMalloc((void**)&pointmass_host.z, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.vz, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.az, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.z, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.vz, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.az, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feedback_az, memorySizeForPointmasses));
#endif
#endif
	cudaVerify(hipHostMalloc((void**)&pointmass_host.rmin, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.rmax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.rmin, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.rmax, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.m, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.m, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.feels_particles, integermemorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feels_particles, integermemorySizeForPointmasses));
#endif

#if MORE_OUTPUT
	cudaVerify(hipHostMalloc((void**)&p_host.p_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.p_max, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.rho_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.rho_max, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.e_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.e_max, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.cs_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.cs_max, memorySizeForParticles));
#endif

	cudaVerify(hipHostMalloc((void**)&p_host.noi, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&p_host.depth, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&interactions_host, memorySizeForInteractions*MAX_NUM_INTERACTIONS));
	cudaVerify(hipHostMalloc((void**)&p_host.materialId, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&childList_host, memorySizeForChildren));

#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipMalloc((void**)&p_device.muijmax, memorySizeForParticles));
#endif

#if (NAVIER_STOKES || BALSARA_SWITCH || INVISCID_SPH || INTEGRATE_ENERGY)
	cudaVerify(hipMalloc((void**)&p_device.divv, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.curlv, memorySizeForParticles*DIM));
#endif

#if INVISCID_SPH
	cudaVerify(hipMalloc((void**)&p_device.beta, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.beta_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.divv_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dbetadt, memorySizeForParticles));
#endif

#if TENSORIAL_CORRECTION
	cudaVerify(hipMalloc((void**)&p_device.tensorialCorrectionMatrix, memorySizeForStress));
	//cudaVerify(hipMalloc((void**)&p_device.tensorialCorrectiondWdrr, MAX_NUM_INTERACTIONS * maxNumberOfParticles * sizeof(double)));
#endif

#if SHEPARD_CORRECTION
	cudaVerify(hipMalloc((void**)&p_device.shepard_correction, memorySizeForParticles));
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipHostMalloc((void**)&p_host.dedt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dedt, memorySizeForParticles));
#endif

#if DISPH
    cudaVerify(hipHostMalloc((void**)&p_host.dUdt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.dUdt, memorySizeForParticles));
#endif

	cudaVerify(hipHostMalloc((void**)&p_host.drhodt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.drhodt, memorySizeForParticles));

#if SOLID
	cudaVerify(hipHostMalloc((void**)&p_host.S, memorySizeForStress));
	cudaVerify(hipHostMalloc((void**)&p_host.dSdt, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.S, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.dSdt, memorySizeForStress));
	cudaVerify(hipHostMalloc((void**)&p_host.local_strain, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.local_strain, memorySizeForParticles));
	cudaVerify(hipMalloc((void**) &p_device.sigma, memorySizeForStress));
#endif

#if NAVIER_STOKES
	cudaVerify(hipHostMalloc((void**)&p_host.Tshear, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.Tshear, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.eta, memorySizeForParticles));
#endif

#if ARTIFICIAL_STRESS
	cudaVerify(hipMalloc((void**) &p_device.R, memorySizeForStress));
#endif

#if JC_PLASTICITY
	cudaVerify(hipHostMalloc((void**)&p_host.ep, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.T, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.ep, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.edotp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.T, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dTdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.jc_f, memorySizeForParticles));
#endif

#if FRAGMENTATION
	memorySizeForActivationThreshold = maxNumberOfParticles * MAX_NUM_FLAWS * sizeof(double);
	cudaVerify(hipHostMalloc((void**)&p_host.d, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.dddt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.d, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.damage_total, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dddt, memorySizeForParticles));

	cudaVerify(hipHostMalloc((void**)&p_host.numFlaws, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.numFlaws, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&p_host.numActiveFlaws, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.numActiveFlaws, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&p_host.flaws, memorySizeForActivationThreshold));
	cudaVerify(hipMalloc((void**)&p_device.flaws, memorySizeForActivationThreshold));
# if PALPHA_POROSITY
    cudaVerify(hipHostMalloc((void**)&p_host.damage_porjutzi, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.ddamage_porjutzidt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.damage_porjutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.ddamage_porjutzidt, memorySizeForParticles));
# endif
#endif

	cudaVerify(hipMalloc((void**)&p_device.h0, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.h0, memorySizeForParticles));

#if GHOST_BOUNDARIES
	cudaVerify(hipMalloc((void**)&p_device.real_partner, memorySizeForInteractions));
#endif

#if PALPHA_POROSITY
	cudaVerify(hipHostMalloc((void**)&p_host.alpha_jutzi, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.alpha_jutzi_old, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.pold, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.dalphadt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.pold, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.alpha_jutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.alpha_jutzi_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dalphadt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dalphadp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dalphadrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.f, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.delpdelrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.delpdele, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs_old, memorySizeForParticles));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipHostMalloc((void**)&p_host.compressive_strength, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.tensile_strength, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.shear_strength, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho_0prime, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho_c_plus, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho_c_minus, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.K, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.flag_rho_0prime, memorySizeForInteractions));
    cudaVerify(hipHostMalloc((void**)&p_host.flag_plastic, memorySizeForInteractions));
    cudaVerify(hipMalloc((void**)&p_device.compressive_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.tensile_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.shear_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.K, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.rho_0prime, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.rho_c_plus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.rho_c_minus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.flag_rho_0prime, memorySizeForInteractions));
    cudaVerify(hipMalloc((void**)&p_device.flag_plastic, memorySizeForInteractions));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipHostMalloc((void**)&p_host.alpha_epspor, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.epsilon_v, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.alpha_epspor, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.dalpha_epspordt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.epsilon_v, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.depsilon_vdt, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&p_device.x, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.g_x, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_local_cellsize, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.vx, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dxdt, memorySizeForParticles));

#if DIM > 1
	cudaVerify(hipMalloc((void**)&p_device.y, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.g_y, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.vy, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dydt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.y0, memorySizeForTree));
    cudaVerify(hipMalloc((void**)&p_device.vy0, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vy0, memorySizeForTree));
#endif

    cudaVerify(hipMalloc((void**)&p_device.x0, memorySizeForTree));
    cudaVerify(hipMalloc((void**)&p_device.vx0, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vx0, memorySizeForTree));
#if DIM > 2
    cudaVerify(hipMalloc((void**)&p_device.z0, memorySizeForTree));
    cudaVerify(hipMalloc((void**)&p_device.vz0, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vz0, memorySizeForTree));
#endif

#if XSPH
	cudaVerify(hipMalloc((void**)&p_device.xsphvx, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&p_device.xsphvy, memorySizeForParticles));
#endif
#endif
	cudaVerify(hipMalloc((void**)&p_device.ax, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_ax, memorySizeForParticles));

#if DIM > 1
	cudaVerify(hipMalloc((void**)&p_device.ay, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_ay, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&p_device.m, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.h, memorySizeForParticles));

#if INTEGRATE_SML
	cudaVerify(hipMalloc((void**)&p_device.dhdt, memorySizeForParticles));
#endif

#if SML_CORRECTION
	cudaVerify(hipMalloc((void**)&p_device.sml_omega, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&p_device.rho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.p, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.e, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.depth, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.noi, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.materialId, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.materialId0, memorySizeForInteractions));

#if DISPH
    cudaVerify(hipMalloc((void**)&p_device.q, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.U, memorySizeForParticles));
#endif

#if MORE_OUTPUT
	cudaVerify(hipMalloc((void**)&p_device.p_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.p_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.rho_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.rho_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.e_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.e_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs_max, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&interactions, memorySizeForInteractions*MAX_NUM_INTERACTIONS));
	cudaVerify(hipMalloc((void**)&childListd, memorySizeForChildren));
#if DIM > 2
	cudaVerify(hipMalloc((void**)&p_device.z, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.g_z, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dzdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.vz, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.az, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_az, memorySizeForParticles));
#if XSPH
	cudaVerify(hipMalloc((void**)&p_device.xsphvz, memorySizeForParticles));
#endif
#endif

	cudaVerify(hipMemset(p_device.ax, 0, memorySizeForParticles));
	cudaVerify(hipMemset(p_device.g_ax, 0, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMemset(p_device.ay, 0, memorySizeForParticles));
	cudaVerify(hipMemset(p_device.g_ay, 0, memorySizeForParticles));
#endif
#if DIM > 2
	cudaVerify(hipMemset(p_device.az, 0, memorySizeForParticles));
	cudaVerify(hipMemset(p_device.g_az, 0, memorySizeForParticles));
#endif

    return rc;
}



int copy_particle_data_to_device()
{
    int rc = 0;
	// copy particle data to device
	if (param.verbose) printf("copying particle data to device ...\n");

	cudaVerify(hipMemcpy(p_device.x0, p_host.x, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.x, p_host.x, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vx, p_host.vx, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vx0, p_host.vx0, memorySizeForParticles, hipMemcpyHostToDevice));
#if DIM > 1
	cudaVerify(hipMemcpy(p_device.y0, p_host.y, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.y, p_host.y, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vy, p_host.vy, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vy0, p_host.vy0, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if DIM > 2
	cudaVerify(hipMemcpy(p_device.z0, p_host.z, memorySizeForTree, hipMemcpyHostToDevice));
#endif

#if GRAVITATING_POINT_MASSES
	cudaVerify(hipMemcpy(pointmass_device.x, pointmass_host.x, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.vx, pointmass_host.vx, memorySizeForPointmasses, hipMemcpyHostToDevice));
# if DIM > 1
	cudaVerify(hipMemcpy(pointmass_device.y, pointmass_host.y, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.vy, pointmass_host.vy, memorySizeForPointmasses, hipMemcpyHostToDevice));
#  if DIM > 2
	cudaVerify(hipMemcpy(pointmass_device.z, pointmass_host.z, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.vz, pointmass_host.vz, memorySizeForPointmasses, hipMemcpyHostToDevice));
#  endif
# endif
	cudaVerify(hipMemcpy(pointmass_device.rmin, pointmass_host.rmin, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.rmax, pointmass_host.rmax, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.m, pointmass_host.m, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.feels_particles, pointmass_host.feels_particles, integermemorySizeForPointmasses, hipMemcpyHostToDevice));
#endif

	cudaVerify(hipMemcpy(p_device.h, p_host.h, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.cs, p_host.cs, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.m, p_host.m, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.rho, p_host.rho, memorySizeForParticles, hipMemcpyHostToDevice));

#if DISPH
    cudaVerify(hipMemcpy(p_device.q, p_host.q, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.U, p_host.U, memorySizeForParticles, hipMemcpyHostToDevice));
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipMemcpy(p_device.e, p_host.e, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if SOLID
	cudaVerify(hipMemcpy(p_device.S, p_host.S, memorySizeForStress, hipMemcpyHostToDevice));
#endif
#if NAVIER_STOKES
	cudaVerify(hipMemcpy(p_device.Tshear, p_host.Tshear, memorySizeForStress, hipMemcpyHostToDevice));
#endif
#if PALPHA_POROSITY
	cudaVerify(hipMemcpy(p_device.alpha_jutzi, p_host.alpha_jutzi, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.alpha_jutzi_old, p_host.alpha_jutzi_old, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.p, p_host.p, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.pold, p_host.pold, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if MORE_OUTPUT
    cudaVerify(hipMemcpy(p_device.p_min, p_host.p_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.p_max, p_host.p_max, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_min, p_host.rho_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_max, p_host.rho_max, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.e_min, p_host.e_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.e_max, p_host.e_max, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.cs_min, p_host.cs_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.cs_max, p_host.cs_max, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if SIRONO_POROSITY
    cudaVerify(hipMemcpy(p_device.compressive_strength, p_host.compressive_strength, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.tensile_strength, p_host.tensile_strength, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.shear_strength, p_host.shear_strength, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_0prime, p_host.rho_0prime, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_c_plus, p_host.rho_c_plus, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_c_minus, p_host.rho_c_minus, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.K, p_host.K, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.flag_rho_0prime, p_host.flag_rho_0prime, memorySizeForInteractions, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.flag_plastic, p_host.flag_plastic, memorySizeForInteractions, hipMemcpyHostToDevice));
#endif
#if EPSALPHA_POROSITY
    cudaVerify(hipMemcpy(p_device.alpha_epspor, p_host.alpha_epspor, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.epsilon_v, p_host.epsilon_v, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
    cudaVerify(hipMemcpy(p_device.h0, p_host.h0, memorySizeForParticles, hipMemcpyHostToDevice));
#if JC_PLASTICITY
	cudaVerify(hipMemcpy(p_device.ep, p_host.ep, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.T, p_host.T, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if FRAGMENTATION
	cudaVerify(hipMemcpy(p_device.d, p_host.d, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.numFlaws, p_host.numFlaws, memorySizeForInteractions, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.numActiveFlaws, p_host.numActiveFlaws, memorySizeForInteractions, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.flaws, p_host.flaws, memorySizeForActivationThreshold, hipMemcpyHostToDevice));
#if PALPHA_POROSITY
    cudaVerify(hipMemcpy(p_device.damage_porjutzi, p_host.damage_porjutzi, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#endif
	cudaVerify(hipMemcpy(p_device.noi, p_host.noi, memorySizeForInteractions, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.materialId, p_host.materialId, memorySizeForInteractions, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.materialId0, p_host.materialId, memorySizeForInteractions, hipMemcpyHostToDevice));
#if DIM > 2
	cudaVerify(hipMemcpy(p_device.z, p_host.z, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vz, p_host.vz, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
	cudaVerify(hipMemset((void *) childListd, -1, memorySizeForChildren));

    return rc;
}



int free_memory()
{

    int rc = 0;
	// free device memory
	if (param.verbose) printf("freeing device memory...\n");
	cudaVerify(hipFree(p_device.x));
	cudaVerify(hipFree(p_device.g_x));
	cudaVerify(hipFree(p_device.g_local_cellsize));
	cudaVerify(hipFree(p_device.depth));
	cudaVerify(hipFree(p_device.x0));
	cudaVerify(hipFree(p_device.dxdt));
	cudaVerify(hipFree(p_device.vx));
	cudaVerify(hipFree(p_device.vx0));
	cudaVerify(hipHostFree(p_host.vx0));
	cudaVerify(hipFree(p_device.ax));
	cudaVerify(hipFree(p_device.g_ax));
	cudaVerify(hipFree(p_device.m));

#if DIM > 1
	cudaVerify(hipFree(p_device.vy0));
	cudaVerify(hipHostFree(p_host.vy0));
#if DIM > 2
	cudaVerify(hipFree(p_device.vz0));
	cudaVerify(hipHostFree(p_host.vz0));
#endif
#endif
#if DIM > 1
	cudaVerify(hipFree(p_device.y));
	cudaVerify(hipFree(p_device.g_y));
	cudaVerify(hipFree(p_device.y0));
	cudaVerify(hipFree(p_device.vy));
	cudaVerify(hipFree(p_device.dydt));
	cudaVerify(hipFree(p_device.ay));
	cudaVerify(hipFree(p_device.g_ay));
#endif

#if GRAVITATING_POINT_MASSES
	cudaVerify(hipFree(pointmass_device.x));
	cudaVerify(hipFree(pointmass_device.vx));
	cudaVerify(hipFree(pointmass_device.ax));
	cudaVerify(hipFree(pointmass_device.feedback_ax));
# if DIM > 1
	cudaVerify(hipFree(pointmass_device.y));
	cudaVerify(hipFree(pointmass_device.vy));
	cudaVerify(hipFree(pointmass_device.ay));
	cudaVerify(hipFree(pointmass_device.feedback_ay));
#  if DIM > 2
	cudaVerify(hipFree(pointmass_device.z));
	cudaVerify(hipFree(pointmass_device.vz));
	cudaVerify(hipFree(pointmass_device.az));
	cudaVerify(hipFree(pointmass_device.feedback_az));
#  endif
# endif
	cudaVerify(hipFree(pointmass_device.m));
	cudaVerify(hipFree(pointmass_device.feels_particles));
	cudaVerify(hipFree(pointmass_device.rmin));
	cudaVerify(hipFree(pointmass_device.rmax));

	cudaVerify(hipHostFree(pointmass_host.x));
	cudaVerify(hipHostFree(pointmass_host.vx));
	cudaVerify(hipHostFree(pointmass_host.ax));
# if DIM > 1
	cudaVerify(hipHostFree(pointmass_host.y));
	cudaVerify(hipHostFree(pointmass_host.vy));
	cudaVerify(hipHostFree(pointmass_host.ay));
#  if DIM > 2
	cudaVerify(hipHostFree(pointmass_host.z));
	cudaVerify(hipHostFree(pointmass_host.vz));
	cudaVerify(hipHostFree(pointmass_host.az));
#  endif
# endif
	cudaVerify(hipHostFree(pointmass_host.m));
	cudaVerify(hipHostFree(pointmass_host.feels_particles));
	cudaVerify(hipHostFree(pointmass_host.rmin));
	cudaVerify(hipHostFree(pointmass_host.rmax));
#endif

#if XSPH
	cudaVerify(hipFree(p_device.xsphvx));
#if DIM > 1
	cudaVerify(hipFree(p_device.xsphvy));
#endif
#endif
	cudaVerify(hipFree(p_device.h));
	cudaVerify(hipFree(p_device.rho));
	cudaVerify(hipFree(p_device.p));
	cudaVerify(hipFree(p_device.e));
	cudaVerify(hipFree(p_device.cs));
	cudaVerify(hipFree(p_device.noi));

#if DISPH
    cudaVerify(hipFree(p_device.q));
    cudaVerify(hipFree(p_device.U));
#endif

#if MORE_OUTPUT
	cudaVerify(hipFree(p_device.p_min));
    cudaVerify(hipFree(p_device.p_max));
    cudaVerify(hipFree(p_device.rho_min));
    cudaVerify(hipFree(p_device.rho_max));
	cudaVerify(hipFree(p_device.e_min));
    cudaVerify(hipFree(p_device.e_max));
    cudaVerify(hipFree(p_device.cs_min));
    cudaVerify(hipFree(p_device.cs_max));
#endif
#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipFree(p_device.muijmax));
#endif
#if INVISCID_SPH
	cudaVerify(hipFree(p_device.beta));
	cudaVerify(hipFree(p_device.beta_old));
	cudaVerify(hipFree(p_device.divv_old));
#endif
	cudaVerify(hipFree(interactions));
	cudaVerify(hipFree(p_device.materialId));
	cudaVerify(hipFree(p_device.materialId0));
	cudaVerify(hipFree(childListd));
#if DIM > 2
	cudaVerify(hipFree(p_device.z));
	cudaVerify(hipFree(p_device.g_z));
	cudaVerify(hipFree(p_device.z0));
	cudaVerify(hipFree(p_device.dzdt));
	cudaVerify(hipFree(p_device.vz));
#if XSPH
	cudaVerify(hipFree(p_device.xsphvz));
#endif
	cudaVerify(hipFree(p_device.az));
	cudaVerify(hipFree(p_device.g_az));
#endif

#if TENSORIAL_CORRECTION
	cudaVerify(hipFree(p_device.tensorialCorrectionMatrix));
	//cudaVerify(hipFree(p_device.tensorialCorrectiondWdrr));
#endif

#if SHEPARD_CORRECTION
	cudaVerify(hipFree(p_device.shepard_correction));
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipHostFree(p_host.dedt));
	cudaVerify(hipFree(p_device.dedt));
#endif

#if DISPH
    cudaVerify(hipHostFree(p_host.dUdt));
    cudaVerify(hipFree(p_device.dUdt));
#endif

	cudaVerify(hipHostFree(p_host.drhodt));
	cudaVerify(hipFree(p_device.drhodt));

#if INTEGRATE_SML
	cudaVerify(hipFree(p_device.dhdt));
#endif
#if SML_CORRECTION
	cudaVerify(hipFree(p_device.sml_omega));
#endif

#if NAVIER_STOKES
	cudaVerify(hipFree(p_device.Tshear));
	cudaVerify(hipHostFree(p_host.Tshear));
	cudaVerify(hipFree(p_device.eta));
#endif
#if SOLID
	cudaVerify(hipFree(p_device.S));
	cudaVerify(hipFree(p_device.dSdt));
	cudaVerify(hipHostFree(p_host.S));
	cudaVerify(hipHostFree(p_host.dSdt));
	cudaVerify(hipFree(p_device.local_strain));
	cudaVerify(hipHostFree(p_host.local_strain));
	cudaVerify(hipFree(p_device.sigma));
#endif
#if ARTIFICIAL_STRESS
	cudaVerify(hipFree(p_device.R));
#endif

#if JC_PLASTICITY
	cudaVerify(hipFree(p_device.ep));
	cudaVerify(hipFree(p_device.edotp));
	cudaVerify(hipFree(p_device.T));
	cudaVerify(hipFree(p_device.dTdt));
	cudaVerify(hipFree(p_device.jc_f));
#endif

#if GHOST_BOUNDARIES
	cudaVerify(hipFree(p_device.real_partner));
#endif

#if FRAGMENTATION
	cudaVerify(hipHostFree(p_host.d));
	cudaVerify(hipFree(p_device.d));
	cudaVerify(hipFree(p_device.damage_total));
	cudaVerify(hipFree(p_device.dddt));
	cudaVerify(hipHostFree(p_host.dddt));
	cudaVerify(hipHostFree(p_host.numFlaws));
	cudaVerify(hipFree(p_device.numFlaws));
	cudaVerify(hipHostFree(p_host.numActiveFlaws));
	cudaVerify(hipFree(p_device.numActiveFlaws));
	cudaVerify(hipHostFree(p_host.flaws));
	cudaVerify(hipFree(p_device.flaws));
#if PALPHA_POROSITY
	cudaVerify(hipFree(p_device.damage_porjutzi));
	cudaVerify(hipFree(p_device.cs_old));
	cudaVerify(hipFree(p_device.ddamage_porjutzidt));
#endif
#endif


#if PALPHA_POROSITY
	cudaVerify(hipFree(p_device.alpha_jutzi));
	cudaVerify(hipFree(p_device.alpha_jutzi_old));
	cudaVerify(hipFree(p_device.pold));
	cudaVerify(hipFree(p_device.dalphadt));
	cudaVerify(hipFree(p_device.dalphadp));
	cudaVerify(hipFree(p_device.dp));
	cudaVerify(hipFree(p_device.dalphadrho));
	cudaVerify(hipFree(p_device.f));
	cudaVerify(hipFree(p_device.delpdelrho));
	cudaVerify(hipFree(p_device.delpdele));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipFree(p_device.compressive_strength));
    cudaVerify(hipFree(p_device.tensile_strength));
    cudaVerify(hipFree(p_device.shear_strength));
    cudaVerify(hipFree(p_device.K));
    cudaVerify(hipFree(p_device.rho_0prime));
    cudaVerify(hipFree(p_device.rho_c_plus));
    cudaVerify(hipFree(p_device.rho_c_minus));
    cudaVerify(hipFree(p_device.flag_rho_0prime));
    cudaVerify(hipFree(p_device.flag_plastic));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipFree(p_device.alpha_epspor));
    cudaVerify(hipFree(p_device.dalpha_epspordt));
    cudaVerify(hipFree(p_device.epsilon_v));
    cudaVerify(hipFree(p_device.depsilon_vdt));
#endif

	// free host memory
	if (param.verbose) printf("freeing host memory...\n");
	cudaVerify(hipHostFree(p_host.x));
	cudaVerify(hipHostFree(p_host.vx));
	cudaVerify(hipHostFree(p_host.ax));
    cudaVerify(hipHostFree(p_host.g_ax));
#if DIM > 1
	cudaVerify(hipHostFree(p_host.y));
	cudaVerify(hipHostFree(p_host.vy));
	cudaVerify(hipHostFree(p_host.ay));
    cudaVerify(hipHostFree(p_host.g_ay));
#endif
	cudaVerify(hipHostFree(p_host.m));
	cudaVerify(hipHostFree(p_host.h));
	cudaVerify(hipHostFree(p_host.rho));
	cudaVerify(hipHostFree(p_host.p));
	cudaVerify(hipHostFree(p_host.e));
	cudaVerify(hipHostFree(p_host.cs));
	cudaVerify(hipHostFree(p_host.noi));
	cudaVerify(hipHostFree(interactions_host));
	cudaVerify(hipHostFree(p_host.depth));
	cudaVerify(hipHostFree(p_host.materialId));
	cudaVerify(hipHostFree(childList_host));

#if DISPH
    cudaVerify(hipHostFree(p_host.q));
    cudaVerify(hipHostFree(p_host.U));
#endif

#if MORE_OUTPUT
	cudaVerify(hipHostFree(p_host.p_min));
	cudaVerify(hipHostFree(p_host.p_max));
	cudaVerify(hipHostFree(p_host.rho_min));
	cudaVerify(hipHostFree(p_host.rho_max));
	cudaVerify(hipHostFree(p_host.e_min));
	cudaVerify(hipHostFree(p_host.e_max));
	cudaVerify(hipHostFree(p_host.cs_min));
	cudaVerify(hipHostFree(p_host.cs_max));
#endif
#if INVISCID_SPH
	cudaVerify(hipHostFree(p_host.beta));
	cudaVerify(hipHostFree(p_host.beta_old));
	cudaVerify(hipHostFree(p_host.divv_old));
#endif
#if PALPHA_POROSITY
	cudaVerify(hipHostFree(p_host.alpha_jutzi));
	cudaVerify(hipHostFree(p_host.alpha_jutzi_old));
	cudaVerify(hipHostFree(p_host.dalphadt));
	cudaVerify(hipHostFree(p_host.pold));
#if FRAGMENTATION
    cudaVerify(hipHostFree(p_host.damage_porjutzi));
    cudaVerify(hipHostFree(p_host.ddamage_porjutzidt));
#endif
#endif

#if SIRONO_POROSITY
    cudaVerify(hipHostFree(p_host.compressive_strength));
    cudaVerify(hipHostFree(p_host.tensile_strength));
    cudaVerify(hipHostFree(p_host.shear_strength));
    cudaVerify(hipHostFree(p_host.rho_0prime));
    cudaVerify(hipHostFree(p_host.rho_c_plus));
    cudaVerify(hipHostFree(p_host.rho_c_minus));
    cudaVerify(hipHostFree(p_host.K));
    cudaVerify(hipHostFree(p_host.flag_rho_0prime));
    cudaVerify(hipHostFree(p_host.flag_plastic));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipHostFree(p_host.alpha_epspor));
    cudaVerify(hipHostFree(p_host.epsilon_v));
#endif

#if JC_PLASTICITY
	cudaVerify(hipHostFree(p_host.ep));
	cudaVerify(hipHostFree(p_host.T));
#endif
#if DIM > 2
	cudaVerify(hipHostFree(p_host.z));
	cudaVerify(hipHostFree(p_host.vz));
	cudaVerify(hipHostFree(p_host.az));
    cudaVerify(hipHostFree(p_host.g_az));
#endif

    if (param.verbose)
        printf("freeing ANEOS (global) host memory...\n");
    free_aneos_memory();

    return rc;
}
