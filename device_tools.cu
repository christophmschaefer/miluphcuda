#include "hip/hip_runtime.h"
/**
 * @author      Daniel Thun and Christoph Schaefer
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */
#include <stdio.h>
#include "device_tools.h"


/* 
        device informations
        authors: Daniel Thun and Christoph Schaefer
        mainly taken from cuda samples
*/



/* ********************************************************************* */
inline int _ConvertSMVer2Cores(int major, int minor)
/*!
 *  Helper function to calculate the number of CUDA core.
 *  Taken from cuda_samples/common/inc/hip/hip_runtime_api.h
 *********************************************************************** */
{
    /* Defines for GPU Architecture types (using the SM version to determine the # of cores per SM */
    typedef struct
    {
        int SM; /* 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version */
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, /* Tesla Generation (SM 1.0) G80 class */
        { 0x11,  8 }, /* Tesla Generation (SM 1.1) G8x class */
        { 0x12,  8 }, /* Tesla Generation (SM 1.2) G9x class */
        { 0x13,  8 }, /* Tesla Generation (SM 1.3) GT200 class */
        { 0x20, 32 }, /* Fermi Generation (SM 2.0) GF100 class */
        { 0x21, 48 }, /* Fermi Generation (SM 2.1) GF10x class */
        { 0x30, 192}, /* Kepler Generation (SM 3.0) GK10x class */
        { 0x32, 192}, /* Kepler Generation (SM 3.2) GK10x class */
        { 0x35, 192}, /* Kepler Generation (SM 3.5) GK11x class */
        { 0x37, 192}, /* Kepler Generation (SM 3.7) GK21x class */
        { 0x50, 128}, /* Maxwell Generation (SM 5.0) GM10x class */
        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    /* If we don't find the values, we default use the previous one to run properly */
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}


/* ********************************************************************* */
void printfDeviceInformation(void)
/*!
 *  printfs some basic information about detected CUDA devices. 
 *  Taken from cuda samples/1_Utilities/deviceQuery
 *  
 *********************************************************************** */
{
int i, device_count, driverVersion = 0, runtimeVersion = 0;
struct hipDeviceProp_t prop;

hipGetDeviceCount(&device_count);

    if (device_count == 0)
    {
        printf("\n> No device(s) that support CUDA found!\n");
        exit(1);
    }

    for (i = 0; i < device_count; i++)
    {   
      //  hipSetDevice(i);
        hipGetDeviceProperties(&prop, i);
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);

        printf("\n> General Information for Device %d -- %s\n\n", i, prop.name);
        printf("  CUDA Driver Version:                           %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
        printf("  CUDA Runtime Version:                          %d.%d\n", runtimeVersion/1000,
(runtimeVersion%100)/10);
        printf("  Compute capability:                            %d.%d\n\n", prop.major, prop.minor);


        printf("  Multiprocessors:                               %d\n", prop.multiProcessorCount);
        printf("  CUDA Cores / Multiprocessor:                   %d\n", _ConvertSMVer2Cores(prop.major, prop.minor));
        printf("  Total amount of CUDA Cores:                    %d\n", _ConvertSMVer2Cores(prop.major,
prop.minor)*prop.multiProcessorCount);
        printf("  GPU clock rate:                                %0.f MHz\n\n", prop.clockRate * 1e-3f);

        #if CUDART_VERSION >= 5000
        /* This is supported in CUDA 5.0 (runtime API device properties) */
        printf("  Memory Clock rate:                             %.0f Mhz\n", prop.memoryClockRate * 1e-3f);
        printf("  Memory Bus Width:                              %d-bit\n",   prop.memoryBusWidth);

        if (prop.l2CacheSize)
        {
            printf("  L2 Cache Size:                                 %d bytes\n\n", prop.l2CacheSize);
        }
        #endif

        printf("  Total amount of global memory:                 %.0f MBytes\n", (float)prop.totalGlobalMem/1048576.0f);
        printf("  Total amount of constant memory:               %lu bytes\n", prop.totalConstMem);
        printf("  Total amount of shared memory per block:       %lu bytes\n", prop.sharedMemPerBlock);

        /* if prop.major >= 3 set shared memory bank size to 8 byte */
        if (prop.major >= 3)
        {
            hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
            printf("  Shared memory bank size                        %d bytes\n", 8);
        }
        else
        {
            printf("  Shared memory bank size                        %d bytes\n", 4);
        }

        printf("  Total number of registers available per block: %d\n", prop.regsPerBlock);
        printf("  Warp size:                                     %d\n\n", prop.warpSize);

        printf("  Maximum number of threads per multiprocessor:  %d\n", prop.maxThreadsPerMultiProcessor);
        printf("  Maximum number of threads per block:           %d\n", prop.maxThreadsPerBlock);
        printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n", prop.maxThreadsDim[0],
prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n\n", prop.maxGridSize[0],
prop.maxGridSize[1], prop.maxGridSize[2]);

        printf("  Run time limit on kernels:                     %s\n", prop.kernelExecTimeoutEnabled ? "Yes" : "No");
        printf("  Integrated GPU sharing Host Memory:            %s\n", prop.integrated ? "Yes" : "No");
        printf("  Support host page-locked memory mapping:       %s\n", prop.canMapHostMemory ? "Yes" : "No");
        printf("  Device has ECC support:                        %s\n\n", prop.ECCEnabled ? "Enabled" : "Disabled");
    }
}
