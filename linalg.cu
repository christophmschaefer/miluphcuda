#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */


#include "timeintegration.h"
#include "miluph.h"
#include "parameter.h"
#include "linalg.h"



__device__ void copy_matrix(double src[DIM][DIM], double dst[DIM][DIM])
{
    int i, j;

    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            dst[i][j] = src[i][j];
        }
    }

}

__device__ void transpose_matrix(double m[DIM][DIM])
{
    int i, j;
    double mt[DIM][DIM];
    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            mt[j][i] = m[i][j];
        }
    }
    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            m[i][j] = mt[i][j];
        }
    }
}

// calculates C = A B and stores in C
__device__  void multiply_matrix(double A[DIM][DIM], double B[DIM][DIM], double C[DIM][DIM])
{
    int i, j, k;

    double vprime[DIM][DIM];

    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            vprime[i][j] = 0.0;
        }
    }

    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            for (k = 0; k < DIM; k++) {
                vprime[i][j] += A[i][k]*B[k][j];
            }
        }
    }
    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            C[i][j] = vprime[i][j];
        }
    }

}

__device__ void identity_matrix(double A[DIM][DIM])
{
    int i, j;
    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            A[i][j] = 0.0;
        }
        A[i][i] = 1.0;
    }
}





// returns the indices of the greatest non-diagonal element of M
__device__ int max_Matrix(double M[DIM][DIM], int *e, int *f, double *elmax)
{
    int i, j;
    double max = 0.0;
    int ierror = 1;

    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            if (i == j)
                continue;
            if (fabs(M[i][j]) >= max) {
                max = fabs(M[i][j]);
                *e = i;
                *f = j;
                ierror = 0;
            }
        }
    }
    *elmax = max;
    return ierror;
}


/*
 * help function for the jacobi method
 * returns: M' = A^T M A, and A_ef = s = -A_ef, A_ee = A_ff = c
 */
__device__ void rotate_matrix(volatile double m[DIM][DIM], volatile double c, volatile double s, volatile int e,
volatile int f)
{
    int i, j;
    volatile double mprime[DIM][DIM];

    /* first copy the matrix */
    for (i = 0; i < DIM; i++)
        for (j = 0; j < DIM; j++)
            mprime[i][j] = m[i][j];

    /* now the elements that change */
    mprime[e][e] = c*c*m[e][e] + s*s*m[f][f] - 2*s*c*m[e][f];
    mprime[f][f] = c*c*m[f][f] + s*s*m[e][e] + 2*s*c*m[e][f];
    mprime[e][f] = (c*c-s*s)*m[e][f] + s*c*(m[e][e]-m[f][f]);
    mprime[f][e] = mprime[e][f];

    /* the other elements in columns and rows e, f*/
    /* actually, this is only one in 3D and 0 in 2D */
    for (i = 0; i < DIM; i++) {
        if (i == f || i == e)
            continue;
        mprime[e][i] = c*m[i][e] - s*m[i][f];
        mprime[i][e] = mprime[e][i];
        mprime[f][i] = c*m[i][f] + s*m[i][e];
        mprime[i][f] = mprime[f][i];
    }

    /* set the matrix to the rotated one */
    for (i = 0; i < DIM; i++)
        for (j = 0; j < DIM; j++)
            m[i][j] = mprime[i][j];
}



/*
 * computes all eigenvalues and eigenvectors of the _symmetric_ matrix M
 * using the jacobi method and stores them in eigenvals and the eigenvecs as columns
 * in the transformation matrix v
 *
 * returns the number of iterations
 */
__device__ int calculate_all_eigenvalues(double M[DIM][DIM], double eigenvalues[DIM], double v[DIM][DIM]) {
    int i, j;
    double diagM[DIM][DIM] = {0.0, };
    double c, s, t, thta;
    double A[DIM][DIM];
    double vtmp[DIM][DIM];
    int e, f;
    int error;
    double max = -1e300;
    int nit = 0;
    i = j = e = f = 0;
    c = s = t = thta = 0.0;
    error = 0;

#define EPS_JACOBI 1e-10

    for (i = 0; i < DIM; i++) {
        for (j = 0; j < DIM; j++) {
            diagM[i][j] = M[i][j];
            v[i][j] = 0.0;
        }
        v[i][i] = 1.0;
    }

    do {
        nit++;
        error = max_Matrix(diagM, &e, &f, &max);
        if (error) {
            printf("No maximum element found.\n");
        }
        if (max > 0) {
            // rotate matrix
            thta = (diagM[f][f] - diagM[e][e])/(2*diagM[e][f]);
            if (thta < 0)
                t = -1./(fabs(thta) + sqrt(thta*thta+1));
            else
                t = 1./(fabs(thta) + sqrt(thta*thta+1));
            // the elements of the rotation matrix
            c = 1./(sqrt(t*t+1));
            s = t*c;
            // do diagM' = A^T diagM A
            rotate_matrix(diagM, c, s, e, f);
            identity_matrix(A);
            A[e][e] = c;
            A[f][f] = c;
            A[e][f] = -s;
            A[f][e] = s;
            // calculate the eigenvectors
            multiply_matrix(v, A, vtmp);
            copy_matrix(vtmp, v);
        }
    } while (max > EPS_JACOBI);

    for (i = 0; i < DIM; i++) {
        eigenvalues[i] = diagM[i][i];
    }
    return nit;
}





/*
 * computes the eigenvalues of the _symmetric_ matrix M
 * using the jacobi method
 * returns the greatest eigenvalue
 */
__device__ double calculateMaxEigenvalue(double M[DIM][DIM]) {
    int i, j;
    double diagM[DIM][DIM] = {0.0, };
    double c, s, t, thta;
    int e, f;
    int error;
    double max;
    double max_ev;
    int nit = 0;
    i = j = e = f = 0;
    c = s = t = thta = 0.0;
    max = max_ev = 0;
    error = 0;


#define EPS_JACOBI 1e-10

    for (i = 0; i < DIM; i++)
        for (j = 0; j < DIM; j++)
            diagM[i][j] = M[i][j];

    do {
        nit++;
        error = max_Matrix(diagM, &e, &f, &max);
        if (error) {
            printf("No maximum element found.\n");
        }
        if (max > 0) {
            // rotate matrix
            thta = (diagM[f][f] - diagM[e][e])/(2*diagM[e][f]);
            if (thta < 0)
                t = -1./(fabs(thta) + sqrt(thta*thta+1));
            else
                t = 1./(fabs(thta) + sqrt(thta*thta+1));
            // the elements of the rotation matrix
            c = 1./(sqrt(t*t+1));
            s = t*c;
            // do diagM' = A^T diagM A
            rotate_matrix(diagM, c, s, e, f);
        }
    } while (max > EPS_JACOBI || nit < 5);

    max_ev = diagM[0][0];
    for (i = 1; i < DIM; i++) {
        if (diagM[i][i] > max_ev) {
            max_ev = diagM[i][i];
        }
    }
    return max_ev;
}

__device__ double det2x2(double a, double b, double c, double d) {
    return a*d-c*b;
}

__device__ int invertMatrix(double *m, double *inverted) {
    double det;
#if (DIM == 2)
    double a, b, c, d;
    a = m[0*DIM+0];
    b = m[0*DIM+1];
    c = m[1*DIM+0];
    d = m[1*DIM+1];

    det = det2x2(a,b,c,d);
  //  if (det < 1e-8) return -1;
   // if (det < 1e-10) det = 1e-10;
    det = 1./det;

    inverted[0*DIM+0] = det*d;
    inverted[0*DIM+1] = -det*b;
    inverted[1*DIM+0] = -det*c;
    inverted[1*DIM+1] = det*a;
#elif (DIM == 3)
    det = m[0 * DIM + 0] * (m[1 * DIM + 1] * m[2 * DIM + 2] - m[2 * DIM + 1] * m[1 * DIM + 2])
        - m[0 * DIM + 1] * (m[1 * DIM + 0] * m[2 * DIM + 2] - m[1 * DIM + 2] * m[2 * DIM + 0])
        + m[0 * DIM + 2] * (m[1 * DIM + 0] * m[2 * DIM + 1] - m[1 * DIM + 1] * m[2 * DIM + 0]);

    // inverse determinante

    if (det < 1e-8) return -1;
    det = 1.0 / det;

    inverted[0*DIM+0] = (m[1*DIM+ 1] * m[2*DIM+ 2] - m[2*DIM+ 1] * m[1*DIM+ 2]) * det;
    inverted[0*DIM+1] = (m[0*DIM+ 2] * m[2*DIM+ 1] - m[0*DIM+ 1] * m[2*DIM+ 2]) * det;
    inverted[0*DIM+2] = (m[0*DIM+ 1] * m[1*DIM+ 2] - m[0*DIM+ 2] * m[1*DIM+ 1]) * det;
    inverted[1*DIM+0] = (m[1*DIM+ 2] * m[2*DIM+ 0] - m[1*DIM+ 0] * m[2*DIM+ 2]) * det;
    inverted[1*DIM+1] = (m[0*DIM+ 0] * m[2*DIM+ 2] - m[0*DIM+ 2] * m[2*DIM+ 0]) * det;
    inverted[1*DIM+2] = (m[1*DIM+ 0] * m[0*DIM+ 2] - m[0*DIM+ 0] * m[1*DIM+ 2]) * det;
    inverted[2*DIM+0] = (m[1*DIM+ 0] * m[2*DIM+ 1] - m[2*DIM+ 0] * m[1*DIM+ 1]) * det;
    inverted[2*DIM+1] = (m[2*DIM+ 0] * m[0*DIM+ 1] - m[0*DIM+ 0] * m[2*DIM+ 1]) * det;
    inverted[2*DIM+2] = (m[0*DIM+ 0] * m[1*DIM+ 1] - m[1*DIM+ 0] * m[0*DIM+ 1]) * det;
#endif

    return 1;
}
